#include "hip/hip_runtime.h"
#include "helper.h"
#include <sys/time.h>
#include <iostream>
// time stuff 
double start_timer()
{
   struct timeval tp;
   gettimeofday(&tp,NULL);
   return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}

void print_time(double start, const char *  s)
{
    double time = start_timer() - start;
    printf("%s took %lf time \n", s, time) ;      
}
// compare stuff 
void compare_float(float x, float y)
{
    if(x != y)
    {
        printf("x = %f != y = %f \n", x, y) ; 
    }
} 
void compare_int(int x, int y) 
{
    if(x != y)
    {
        printf("x = %i != y = %i \n", x, y) ; 
    }
} 

// make random data 
void make_rand_vector(int dim, des_t &vec)
{
     for (size_t i = 0; i < dim; i++)
        {
            vec[i] = static_cast<float>(rand())  ;
    
        }
        float sum = 0 ; 
        for (size_t i = 0; i < dim; i++)
        {
           sum += vec[i]*vec[i]  ; 
        }
        sum = sqrtf(sum)  ; 
        for (size_t i = 0; i < dim; i++)
        {
            vec[i] /= sum ;  
        } 
}

void make_rand_vec_array(int dim, int size, des_t *array)
{
    des_t *arr = (des_t *)array;
    for (size_t i = 0; i < size; i++)
    {
        make_rand_vector(dim, arr[i]);
    }
}