#include "hip/hip_runtime.h"
#include "knn_brute.h"
#include "lsh_h.h"
#include "hip/hip_runtime_api.h"
#include <iostream>
#include <string>
#include <math.h>
#include <sys/time.h>
#include "helper.h"

#include "hipblas.h"

#include <cstring>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
void make_rand_vector(int dim, des_t_f &vec);
void make_rand_vec_array(int dim, int size, des_t_f *array);

void test_float(); 
void test_half_float(); 
#include <iostream>
#include <fstream>
#include <vector>
#include <functional>

int main(int argc, char *argv[])
{
    test_float();
    //test_half_float() ; 
    return 0;
}

void test_float()
{
    int dim = 128;
    int size_q = 10000;
    int size_r = 1000000 ;

    des_t_f *q_points;
    des_t_f *r_points;
    
    des_t_f *gpu_q_points;
    des_t_f *gpu_r_points;

    float4 *sorted_lsh;
    float4 *sorted_2nn;
    //make streams/ handels 
    int stream_n = 2 ; 
    hipblasHandle_t handle ;  
    hipblasCreate(&handle); 

    uint32_t * matches_lsh ; 
    hipMallocManaged((void **)&matches_lsh, size_q * sizeof(uint32_t));
    uint32_t * matches_brute ; 
    hipMallocManaged((void **)&matches_brute, size_q * sizeof(uint32_t));
    
    hipHostMalloc((void **)&q_points, size_q * sizeof(des_t_f));
    hipHostMalloc((void **)&r_points, size_r * sizeof(des_t_f));
 
    hipMalloc((void **)&gpu_q_points, size_q * sizeof(des_t_f));
    hipMalloc((void **)&gpu_r_points, size_r * sizeof(des_t_f));

    //output arrays dist and index of dist for 2nn 
  //  hipMallocManaged((void **)&sorted_lsh, size_q * sizeof(float4));
   // hipMallocManaged((void **)&sorted_2nn, size_q * sizeof(float4));

    make_rand_vec_array(dim, size_q, q_points);
    make_rand_vec_array(dim, size_r, r_points);

    hipMemcpy(gpu_q_points, q_points, size_q * sizeof(des_t_f), hipMemcpyHostToDevice) ; 
    hipMemcpy(gpu_r_points, r_points, size_r * sizeof(des_t_f), hipMemcpyHostToDevice) ; 
    //   hipProfilerStart();
   // lsh_test(gpu_q_points, gpu_r_points, size_q, size_r, sorted_lsh, 25, 20, 0, handle);
    double s = start_timer() ; 
   //// printf("brute needs to compare %zu points \n", size_q * size_r ) ; 
   //// //host_brute(q_points,r_points,size_q,size_r, sorted_lsh) ;
   // cublas_2nn_f(gpu_q_points,gpu_r_points,size_q,size_r, sorted_2nn, handle) ;
    cublas_2nn_sift(gpu_q_points, gpu_r_points, 1, size_q, size_r, matches_brute, 1, handle, stream_n); 
    hipDeviceSynchronize() ;
   //device_brute(gpu_q_points,gpu_r_points,size_q,size_r, sorted_2nn) ;
   print_time(s, "cublas burte") ; 
 
     s = start_timer();

    lsh_gpu(gpu_q_points, gpu_r_points, 1, size_q, size_r, matches_lsh, 0.8, handle, 2, 1, 0, 32); 
    
    hipDeviceSynchronize() ;
    //    hipProfilerStop() ;
    // lsh_test(gpu_q_points, gpu_r_points, size_q, size_r, sorted_2nn, 15 , 10, 0, handle[0]);
   print_time(s, "lsh brute"); 

  int failed = 0 ; 
    s = start_timer() ; 
   for (size_t i = 0; i < size_q; i++)
   {
       if(matches_lsh[i]!= matches_brute[i] )
       {
           failed ++ ; 
            // printf("%i, %i \n", (int)matches_brute[i], (int)matches_lsh[i]); 
       }

       //   printf("%i, %i \n", (int)matches_brute[i], (int)matches_lsh[i]); 
   }
   print_time(s, "for loop") ; 
   printf("failed %i \n", failed); 
   // 
   // return ; 
   // // see how many poins lsh got right 
   // for (size_t i = 0; i < size_q; i++)
   // {
   //    // printf("\n") ;
   //    if(sorted_2nn[i].z !=  sorted_lsh[i].z)
   //     {
   //         printf("failed z %f , %f\n", sorted_2nn[i].z, sorted_lsh[i].z ); 
   //         printf("w = %f , %f\n", sorted_2nn[i].w, sorted_lsh[i].w ); 
   //        failed ++ ; 
// //           printf("z is bad \n") ; 
  ////  printf("lsh 1  %f index %f  lsh 2 %f index %f \n", sorted_host[i].x, sorted_host[i].z, sorted_host[i].y,  sorted_host[i].w) ; 
  ////  printf("cpu 1  %f index %f  cpu 2 %f index %f \n", sorted_dev[i].x, sorted_dev[i].z, sorted_dev[i].y,  sorted_dev[i].w) ;
   //     }
   //     if(sorted_2nn[i].w !=  sorted_lsh[i].w)
   //     {
   //         printf("failed w %f , %f\n", sorted_2nn[i].w, sorted_lsh[i].w ); 
   //         printf("z =  %f , %f\n", sorted_2nn[i].z, sorted_lsh[i].z ); 
   //         failed ++ ; 
 ////           printf("w is not good \n"); 
   //     }
   //     
   // }
    //printf("found %i out of %i nn \n",((size_q * 2)- failed),(size_q *2) ) ; 
    hipFree(gpu_q_points); 
    hipFree(gpu_r_points); 
    hipFree(q_points); 
    hipFree(r_points); 
    // hipFree(sorted_2nn); 
    // hipFree(sorted_lsh); 
}


void test_half_float()
{
    int dim = 128;
    int size_q = 10000;
    int size_r = 100000;

    des_t_h *q_points;
    des_t_h *r_points;
    
    des_t_h2 *gpu_q_points;
    des_t_h2 *gpu_r_points;

    uint32_t * matches ; 
    //make streams/ handels 
    int stream_n = 2 ; 
   
   
    
    hipHostMalloc((void **)&q_points, size_q * sizeof(des_t_h2),  hipHostMallocMapped);
    hipHostMalloc((void **)&r_points, size_r * sizeof(des_t_h2),  hipHostMallocMapped);
 
    hipMalloc((void **)&gpu_q_points, size_q * sizeof(des_t_h2));
    hipMalloc((void **)&gpu_r_points, size_r * sizeof(des_t_h2));

    //output arrays dist and index of dist for 2nn 
    hipMallocManaged((void **)&matches, size_q * sizeof(uint32_t));

    make_rand_vec_array_h(dim, size_q, q_points);
    make_rand_vec_array_h(dim, size_r, r_points);

    hipMemcpy(gpu_q_points, q_points, size_q * sizeof(des_t_h2), hipMemcpyHostToDevice) ; 
    hipMemcpy(gpu_r_points, r_points, size_r * sizeof(des_t_h2), hipMemcpyHostToDevice) ; 
    double s = start_timer();
    //   hipProfilerStart();
   // lsh_test(gpu_q_points, gpu_r_points, size_q, size_r, sorted_lsh, 25, 20, 0, handle);
    //device_brute(q_points,r_points,size_q,size_r, sorted_lsh) ;
    hipDeviceSynchronize() ;
    //    hipProfilerStop() ;
    //gpu_lsh(q_points, r_points, size_q, size_r, sorted_host, 4, 4, 2);
    print_time(s, "gpu lsh"); 
    s = start_timer() ; 
 //   cublas_2nn_sift(gpu_q_points, gpu_r_points, 2, size_q, size_r, matches, 0.999, handle, stream_n); 

    hipDeviceSynchronize() ;
    printf("brute needs to compare %zu points \n", size_q * size_r ) ; 
    print_time(s, "gpu brute") ; 
    int failed = 0 ; 
    for (size_t i = 0; i < 10; i++)
    {
        printf("match is %i\n", matches[i]) ; 
    }
    
    return ; 
    // see how many poins lsh got right 
    printf("found %i out of %i nn \n",((size_q * 2)- failed),(size_q *2) ) ; 
    hipFree(gpu_q_points); 
    hipFree(gpu_r_points); 
    hipFree(q_points); 
    hipFree(r_points); 
}

  