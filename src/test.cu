#include "hip/hip_runtime.h"
#include "knn_brute.h"
#include "lsh.h"
#include "hip/hip_runtime_api.h"
#include <iostream>
#include <string>
#include <math.h>
#include <sys/time.h>

void make_rand_vector(int dim, des_t &vec);
void make_rand_vec_array(int dim, int size, des_t *array);
void test();

int main(int argc, char *argv[])
{
    test();
    return 0;
}
void test()
{
    int dim = 128;
    int size_q = 5;
    int size_r = 50;
    des_t *q_points;
    des_t *r_points;

    float4 *sorted_host;
    float4 *sorted_dev;
    //host
    hipMallocManaged((void **)&q_points, size_q * sizeof(des_t));
    hipMallocManaged((void **)&r_points, size_r * sizeof(des_t));

    hipMallocManaged((void **)&sorted_host, size_r * sizeof(float4));
    hipMallocManaged((void **)&sorted_dev, size_r * sizeof(float4));

    //data
    make_rand_vec_array(dim, size_q, q_points);
    make_rand_vec_array(dim, size_r, r_points);

    //   hipProfilerStart();
    //   device_brute(q_points, r_points, size_q, size_r, sorted_dev) ;
   // host_brute(q_points,r_points,size_q,size_r, sorted_dev) ;
    //    hipProfilerStop() ;
    host_lsh(q_points, r_points, size_q, size_r, sorted_host, 5, 1, 3);
   // for (size_t i = 0; i < size_q; i++)
   // {
   //     printf("lsh 1  %f index %f  lsh 2 %f index %f \n", sorted_host[i].x, sorted_host[i].z, sorted_host[i].y,  sorted_host[i].w) ;
   //     printf("cpu 1  %f index %f  cpu 2 %f index %f \n", sorted_dev[i].x, sorted_dev[i].z, sorted_dev[i].y,  sorted_dev[i].w) ;
   //     printf("\n") ;
   // }
}

void make_rand_vector(int dim, des_t &vec)
{
    for (size_t i = 0; i < dim; i++)
    {
        vec[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }
}

void make_rand_vec_array(int dim, int size, des_t *array)
{
    des_t *arr = (des_t *)array;
    for (size_t i = 0; i < size; i++)
    {
        make_rand_vector(dim, arr[i]);
    }
}
