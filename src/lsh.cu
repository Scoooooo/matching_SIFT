#include "hip/hip_runtime.h"
#include <iostream>
#include "hiprand/hiprand_kernel.h"
#include <string>
#include "hipblas.h"
#include "lsh.h"
#include "knn_brute.h"
#include <memory>
#include "helper.h"
#include <vector>
#include <algorithm>
#include <bits/stdc++.h>

using namespace std;
//to do make rand array[l*nbits] 128 long and multyply with q_points
// todo cublas scale data to have 0 as center instead of 0.5 , brute only 2nn  ?
void host_lsh(des_t *q_points, des_t *r_points, int n_q, int n_r, float4 *sorted, int nbits, int l, int max_dist)
{

    des_t *rand_array;
    hipMallocManaged((void **)&rand_array, sizeof(des_t) * nbits * l);
    // make random vectors
    for (size_t i = 0; i < l * nbits; i++)
    {
        make_vec(128, rand_array[i * 128]);
    }
    // make an array of ints with one int for each r_point
    int *code;
    int *index;
    // need a copy to sort using sort
    int *index_copy;
    int *bucket_start;

    hipMallocManaged((void **)&index, sizeof(int) * n_r * l);
    hipMemset(index, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&index_copy, sizeof(int) * n_r * l);
    hipMemset(index_copy, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&bucket_start, (2 << nbits) * sizeof(int));

    for (int i = 0; i < (2 << nbits); i++)
    {
        bucket_start[i] = -1;
    }

    hipMallocManaged((void **)&code, sizeof(int) * n_r * l);
    hipMemset(code, 0, sizeof(int) * n_r * l);

    float *test;
    hipMallocManaged((void **)&test, sizeof(float) * n_r * l);

    // dot all vectors and add the bit to the coresponding int bit for the r points
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_r; ii++)
        {
            //            printf(" %i bucket = ", (ii + i * n_r));
            for (int iii = 0; iii < nbits; iii++)
            {
                float sum = dot(r_points[ii], rand_array[iii + i * nbits]);
                // hipblasSdot(handle, 128, r_points[ii], 1, rand_array[iii + i *nbits], 1, &test[ii + i *n_r]) ;
                if (sum <= 0)
                {
                    code[ii + i * n_r] |= 1UL << iii;
                }
            }
            //           printf(" %i ", code[ii + i * n_r]);
            //           printf("\n \n");
        }
    }

    // make buckets for r points

    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_r; ii++)
        {
            index[ii + i * n_r] = ii;
            index_copy[ii + i * n_r] = ii;
        }
    }

    std::sort(index, index + n_r * l, [&](const int &i, const int &j) -> bool
              { return (code[index_copy[i]] < code[index_copy[j]]); });

    //index
    // code[index[r number ]] -> bucket
    //

    // index[range (bucket start[bucket] ->  [while code [index in r:points]] == bucket)] -> index in r_points of elemtn in bucket
    // r_points[index in r_points ] -> dest vec

    int *helper;
    hipMallocManaged((void **)&helper, sizeof(int) * n_r * l);

    //set bucket start
    dim3 block_size(32, 1, 1);
    dim3 grid_size(((l * n_r) / 32) + 1, 1, 1);
    set_helper<<<grid_size, block_size>>>(helper, code, index);

    hipDeviceSynchronize();
    dim3 block(32, 1, 1);
    dim3 grid(n_r * l, 1, 1);

    //  set_bucket_start<<<grid, block>>>(helper, bucket_start, l, n_r);
    //  hipDeviceSynchronize();
    for (int i = 0; i < n_r * l; i++)
    {
        if (bucket_start[code[index[i]]] == -1)
        {
            bucket_start[code[index[i]]] = i;
        }
    }

    int *code_q;
    hipMallocManaged((void **)&code_q, sizeof(int) * n_r * l);
    hipMemset(code_q, 0, sizeof(int) * n_r * l);
    // dot all q points
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_q; ii++)
        {
            for (int iii = 0; iii < nbits; iii++)
            {
                float sum = dot(q_points[ii], rand_array[iii + i * nbits]);
                if (sum <= 0)
                {
                    code_q[ii + i * n_q] |= 1UL << iii;
                }
            }
        }
    }

    int *buckets;
    hipMallocManaged((void **)&buckets, sizeof(int) * n_q * n_r);
    hipMemset(buckets, 0, sizeof(int) * n_q * n_r);
    // fill buckets
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_q; ii++)
        {
            int bucket = code_q[ii + (i * n_q)];
            int start = bucket_start[bucket];
            int iii = start;
            while ((start != -1) && code[index[iii]] == bucket)
            {
                buckets[ii * n_r + index[iii]] = 1;
                iii++;
            }

            // add from negbouring buckets up to n bits away // n can be given by input but is by defualt 1
            // is there any meaing to adding more than

            // make all buckets with a hamming distance of n

            // 0000
            // 1000 0100 0010 0001
            // 1100 1010 1001
            // 0110 0101
            // 0011

            // 010
            // 011 000 110
            // 001 100
            for (int n = 0; n < max_dist; n++)
            {
                int counters[n + 1];

                for (int q = 0; q < (n + 1); q++)
                {
                    counters[q] = q;
                }

                bool done = false;
                while (!done)
                {
                    int neighbour_bucket = bucket;
                    for (int nn = 0; nn < (n + 1); nn++)
                    {
                        neighbour_bucket ^= 1UL << counters[nn];
                    }
                    // printf("bucket is %i neighbour is %i \n", bucket, neighbour_bucket) ;
                    // we have bucket
                    int start = bucket_start[neighbour_bucket];
                    int iii = start;
                    while ((start != -1) && code[index[iii]] == neighbour_bucket)
                    {
                        buckets[ii * n_r + index[iii]] = 1;
                        iii++;
                    }
                    bool flag = false;
                    int nnn = n;
                    int bits = nbits;
                    while (!flag)
                    {

                        if (((counters[nnn] + 1) >= bits) && nnn == 0)
                        {
                            flag = true;
                            done = true;
                        }
                        else if ((counters[nnn] + 1) < bits)
                        {
                            counters[nnn] += 1;
                            flag = true;
                        }

                        nnn--;
                        bits--;
                    }
                }
            }
            //         printf("\n");
        }
    }

    for (int i = 0; i < n_q; i++)
    {
        sorted[i].w = MAXFLOAT;
        sorted[i].x = MAXFLOAT;
        sorted[i].y = MAXFLOAT;
        sorted[i].z = MAXFLOAT;
        for (int ii = 0; ii < n_r; ii++)
        {
            if (buckets[n_r * i + ii] == 1)
            {
                float dist = host_lenght(r_points[ii], q_points[i]);

                if (dist < sorted[i].x)
                {
                    sorted[i].y = sorted[i].x;
                    sorted[i].x = dist;

                    sorted[i].w = sorted[i].z;
                    sorted[i].z = ii;
                }
                else
                {
                    if (dist < sorted[i].y)
                    {
                        sorted[i].y = dist;
                        sorted[i].w = ii;
                    }
                }
            }
        }
    }

    // brute for each q point in the right bucket
    // for (int i = 0; i < l; i++)
    // {
    //     for (int ii = 0; ii < n_q; ii++)
    //     {
    //         int bucket = code_q[ii + (i * n_q)] ;
    //         int start = bucket_start[bucket] ;
    //         int iii = start ;
    //         while ((start != -1) && code[index[iii]] == bucket)
    //         {
    //             float dist = host_lenght(r_points[index[iii]], q_points[ii] ) ;
    //             if(i == 0)
    //             {
    //                 sorted[ii].w = MAXFLOAT ;
    //                 sorted[ii].x = MAXFLOAT ;
    //                 sorted[ii].y = MAXFLOAT ;
    //                 sorted[ii].z = MAXFLOAT ;
    //             }
    //             if(dist < sorted[ii].x)
    //             {
    //                 sorted[ii].y = sorted[ii].x ;
    //                 sorted[ii].x = dist;
    //
    //                 sorted[ii].w= sorted[ii].z;
    //                 sorted[ii].z= index[iii];
    //             }
    //             else{
    //                 if(dist < sorted[ii].y)
    //                 {
    //                     sorted[ii].w = index[iii] ;
    //                 }
    //             }
    //             iii ++ ;
    //         }
    //     }
    // }
}

void make_vec(int dim, des_t &vec)
{
    for (size_t i = 0; i < dim; i++)
    {
        vec[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }
}

float dot(des_t v1, des_t v2)
{

    float sum = 0.f;
    for (size_t i = 0; i < 128; i++)
    {
        float a = ((float *)v1)[i];
        float b = ((float *)v2)[i];
        float c = a * b;
        sum += c;
    }
    return sum;
}
float test_dot(float *v1, des_t v2)
{
    float sum = 0.f;
    for (size_t i = 0; i < 128; i++)
    {
        float a = ((float *)v1)[(4 * 4) * i];
        float b = ((float *)v2)[i];
        float c = a * b;
        sum += c;
    }
    return sum;
}
void gpu_lsh(des_t *q_points, des_t *r_points, int n_q, int n_r, float4 *sorted, int nbits, int l, int max_dist)
{
    // make random vectors

    float *rand_array;
    hipMallocManaged((void **)&rand_array, sizeof(des_t) * nbits * l);

    // fill array
    uint64_t seed = 123451;
    dim3 grid_size(nbits * l, 1, 1);
    dim3 block_size(128, 1, 1);

    //fill in the rand array
    random_vector<<<grid_size, block_size>>>(seed, rand_array);

    // make an array of ints with one int for each r_point
    float *dot_res;
    int *code;
    int *index;
    // need a copy to sort using sort
    int *index_copy;
    int *bucket_start;

    hipMallocManaged((void **)&index, sizeof(int) * n_r * l);
    hipMemset(index, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&index_copy, sizeof(int) * n_r * l);
    hipMemset(index_copy, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&bucket_start, (2 << nbits) * sizeof(int));

    hipMallocManaged((void **)&dot_res, l * nbits * n_r);

    float a = 1.0f;
    float b = 1.0f;
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, l * nbits, n_r, 128, &a, (float *)rand_array, l * nbits, (float *)r_points, 128, &b, dot_res, l * nbits);
    
    /**
     * rand_arraay = L*nbitsX128
     * r_points = 128 X n_r
     * dot_res = L*nbits X n_r if we were to use cublas  so n_r * L*nbits  
     * dot_res[] 
     **/
  // test there are some difrencece caused by the way the calculation is done  
  //  for (int i = 0; i < n_r; i++)
  //  {
  //      for (int ii = 0; ii < (l * nbits); ii++)
  //      {
  //          compare_float(test_dot((rand_array + ii), r_points[i]), dot_res[i * (l * nbits) + ii]);
  //      }
  //  }
    
}

// kernels for finding the start of each bucket in the index array
// not sure if this is really faster than cpu todo TEST
// sets helper values
__global__ void set_helper(int *helper, int *code, int *index)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    helper[idx] = code[index[idx]];
}

__global__ void set_bucket_start(int *helper, int *bucket_start, int l, int n_r)
{
    for (int i = threadIdx.x; i < (l * n_r); i += 32)
    {

        int v = (helper[i] == blockIdx.x);
        unsigned int mask = __ballot_sync(0xffffffff, v);

        if (__popc(mask) == 0)
            continue;
        int x = __ffs(mask);
        if (threadIdx.x == 0)
        {
            int g = i + x - 1;
            bucket_start[blockIdx.x] = g;
            break;
        }
    }
}

// set the bit
__global__ void dot_set_bit(float *rand, float *points, int *buckets, int size, int nbits, float *test)
{
}

// initialize array to a value
__global__ void initialize(int *array, int value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    array[idx] = value;
}

// may have to make vectors more random ! hmm todo
// makes a random float

__device__ inline float random_float(uint64_t seed, int idx)
{
    hiprandState s;
    hiprand_init(seed + idx, 0, 0, &s);
    return hiprand_uniform(&s);
}
//fills a vector with random floats
__global__ void random_vector(uint64_t seed, float *vec)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    vec[idx] = random_float(seed, idx);
}