#include "hip/hip_runtime.h"
#include <iostream>
#include "hiprand/hiprand_kernel.h"
#include <string>
#include "hipblas.h"
#include "lsh.h"
#include "knn_brute.h"
#include <memory>
#include <vector>
#include <algorithm>
#include <bits/stdc++.h>

using namespace std;

void host_lsh(des_t *q_points, des_t *r_points, int n_q, int n_r, float4 *sorted, int nbits, int l, int max_dist)
{

    des_t *rand_array;
    hipMallocManaged((void **)&rand_array, sizeof(des_t) * nbits * l);
    // make random vectors
    for (size_t i = 0; i < nbits * l; i++)
    {
        make_vec(128, rand_array[i]);
    }
    // make an array of ints with one int for each r_point
    int *code;
    int *index;
    int *bucket_start;
    // need a copy to sort using sort
    int *index_copy;

    hipMallocManaged((void **)&index, sizeof(int) * n_r * l);
    hipMemset(index, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&index_copy, sizeof(int) * n_r * l);
    hipMemset(index_copy, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&bucket_start, 2 << nbits);

    for (int i = 0; i < (2 << nbits); i++)
    {
        bucket_start[i] = -1;
    }

    hipMallocManaged((void **)&code, sizeof(int) * n_r * l);
    hipMemset(code, 0, sizeof(int) * n_r * l);

    float * test ;  
    hipMallocManaged((void **)&test, sizeof(float) * n_r * l);
    hipblasHandle_t handle;
    hipblasCreate(&handle) ; 
    
    // dot all vectors and add the bit to the coresponding int bit for the r points
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_r; ii++)
        {
            //            printf(" %i bucket = ", (ii + i * n_r));
            for (int iii = 0; iii < nbits; iii++)
            {
                float sum = dot(r_points[ii], rand_array[iii + i * nbits]);
               // hipblasSdot(handle, 128, r_points[ii], 1, rand_array[iii + i *nbits], 1, &test[ii + i *n_r]) ; 
                if ( sum<= 0)
                {
                    code[ii + i * n_r] |= 1UL << iii;
                }
            }
            //           printf(" %i ", code[ii + i * n_r]);
            //           printf("\n \n");
        }
    }

    // make buckets for r points

    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_r; ii++)
        {
            index[ii + i * n_r] = ii;
            index_copy[ii + i * n_r] = ii;
        }
    }

    std::sort(index, index + n_r * l, [&](const int &i, const int &j) -> bool
              { return (code[index_copy[i]] < code[index_copy[j]]); });

    //index
    // code[index[r number ]] -> bucket
    //

    // index[range (bucket start[bucket] ->  [while code [index in r:points]] == bucket)] -> index in r_points of elemtn in bucket
    // r_points[index in r_points ] -> dest vec

    int *helper;
    hipMallocManaged((void **)&helper, sizeof(int) * n_r * l);

    //set bucket start
    dim3 block_size(32, 1, 1);
    dim3 grid_size(((l * n_r) / 32) + 1, 1, 1);
    set_helper<<<grid_size, block_size>>>(helper, code, index);

    hipDeviceSynchronize();
    dim3 block(32, 1, 1);
    dim3 grid(n_r * l, 1, 1);

    min_helper<<<grid, block>>>(helper, bucket_start, l, n_r);
    hipDeviceSynchronize();
 //   for (int i = 0; i < n_r * l; i++)
 //   {
 //       if (bucket_start[code[index[i]]] == -1)
 //       {
 //           bucket_start[code[index[i]]] = i;
 //       }
 //  }
   
    int *code_q;
    hipMallocManaged((void **)&code_q, sizeof(int) * n_r * l);
    hipMemset(code_q, 0, sizeof(int) * n_r * l);
    // dot all q points
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_q; ii++)
        {
            for (int iii = 0; iii < nbits; iii++)
            {
                float sum = dot(q_points[ii], rand_array[iii + i * nbits]);
                if (sum <= 0)
                {
                    code_q[ii + i * n_q] |= 1UL << iii;
                }
            }
        }
    }

    int *buckets;
    hipMallocManaged((void **)&buckets, sizeof(int) * n_q * n_r);
    hipMemset(buckets, 0, sizeof(int) * n_q * n_r);
    // fill buckets
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_q; ii++)
        {
            int bucket = code_q[ii + (i * n_q)];
            int start = bucket_start[bucket];
            int iii = start;
            while ((start != -1) && code[index[iii]] == bucket)
            {
                buckets[ii * n_r + index[iii]] = 1;
                iii++;
            }

            // add from negbouring buckets up to n bits away // n can be given by input but is by defualt 1
            // is there any meaing to adding more than

            // make all buckets with a hamming distance of n

            // 0000
            // 1000 0100 0010 0001
            // 1100 1010 1001
            // 0110 0101
            // 0011

            // 010
            // 011 000 110
            // 001 100
            // for (int n = 0; n < max_dist; n++)
            // {
            //     int counters[n + 1] ;

            //     for (int q = 0; q < (n + 1); q++)
            //     {
            //         counters[q] = q ;
            //     }
            //
            //     for (int nn = 0; nn < (n+1); n++)
            //     {
            //         for (int nnn = 0; nnn < (nbits - nn) ; nnn++)
            //         {
            //             for (int nnnn = 0; nnnn < ; nnnn++)
            //             {
            //                 /* code */
            //             }
            //
            //
            //         }
            //
            //     }
            //

            //
            //
            // }

            for (int n = 0; n < max_dist; n++)
            {
                int counters[n + 1];

                for (int q = 0; q < (n + 1); q++)
                {
                    counters[q] = q;
                }

                bool done = false;
                while (!done)
                {
                    int neighbour_bucket = bucket;
                    for (int nn = 0; nn < (n + 1); nn++)
                    {
                        neighbour_bucket ^= 1UL << counters[nn];
                    }
                    // printf("bucket is %i neighbour is %i \n", bucket, neighbour_bucket) ;
                    // we have bucket
                    int start = bucket_start[neighbour_bucket];
                    int iii = start;
                    while ((start != -1) && code[index[iii]] == neighbour_bucket)
                    {
                        buckets[ii * n_r + index[iii]] = 1;
                        iii++;
                    }
                    bool flag = false;
                    int nnn = n;
                    int bits = nbits;
                    while (!flag)
                    {

                        if (((counters[nnn] + 1) >= bits) && nnn == 0)
                        {
                            flag = true;
                            done = true;
                        }
                        else if ((counters[nnn] + 1) < bits)
                        {
                            counters[nnn] += 1;
                            flag = true;
                        }

                        nnn--;
                        bits--;
                    }
                }
            }
            //         printf("\n");
        }
    }

    for (int i = 0; i < n_q; i++)
    {
        sorted[i].w = MAXFLOAT;
        sorted[i].x = MAXFLOAT;
        sorted[i].y = MAXFLOAT;
        sorted[i].z = MAXFLOAT;
        for (int ii = 0; ii < n_r; ii++)
        {
            if (buckets[n_r * i + ii] == 1)
            {
                float dist = host_lenght(r_points[ii], q_points[i]);

                if (dist < sorted[i].x)
                {
                    sorted[i].y = sorted[i].x;
                    sorted[i].x = dist;

                    sorted[i].w = sorted[i].z;
                    sorted[i].z = ii;
                }
                else
                {
                    if (dist < sorted[i].y)
                    {
                        sorted[i].y = dist;
                        sorted[i].w = ii;
                    }
                }
            }
        }
    }

    // brute for each q point in the right bucket
    // for (int i = 0; i < l; i++)
    // {
    //     for (int ii = 0; ii < n_q; ii++)
    //     {
    //         int bucket = code_q[ii + (i * n_q)] ;
    //         int start = bucket_start[bucket] ;
    //         int iii = start ;
    //         while ((start != -1) && code[index[iii]] == bucket)
    //         {
    //             float dist = host_lenght(r_points[index[iii]], q_points[ii] ) ;
    //             if(i == 0)
    //             {
    //                 sorted[ii].w = MAXFLOAT ;
    //                 sorted[ii].x = MAXFLOAT ;
    //                 sorted[ii].y = MAXFLOAT ;
    //                 sorted[ii].z = MAXFLOAT ;
    //             }
    //             if(dist < sorted[ii].x)
    //             {
    //                 sorted[ii].y = sorted[ii].x ;
    //                 sorted[ii].x = dist;
    //
    //                 sorted[ii].w= sorted[ii].z;
    //                 sorted[ii].z= index[iii];
    //             }
    //             else{
    //                 if(dist < sorted[ii].y)
    //                 {
    //                     sorted[ii].w = index[iii] ;
    //                 }
    //             }
    //             iii ++ ;
    //         }
    //     }
    // }
}

void make_vec(int dim, des_t &vec)
{
    for (size_t i = 0; i < dim; i++)
    {
        vec[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }
}

float dot(des_t v1, des_t v2)
{

    float sum = 0.f;
    for (size_t i = 0; i < 128; i++)
    {
        float a = ((float *)v1)[i] - 0.5;
        float b = ((float *)v2)[i] - 0.5;
        float c = a * b;
        sum += c;
    }
    return sum;
}

void gpu_lsh(des_t *q_points, des_t *r_points, int n_q, int n_r, float4 *sorted, int nbits, int l)
{
    // repeat l times !! will lead to comparing the same points multiple times ?

    // make random vectors

    des_t *rand_array;
    hipMallocManaged((void **)&rand_array, sizeof(des_t) * nbits * l);

    // fill array
    uint64_t seed = 9753;
    dim3 grid_size(1, 1, 1);
    dim3 block_size(32, 1, 1);

    //fill in the dist array
    random_vector<<<grid_size, block_size>>>(seed, rand_array);
}

// kernels for finding the start of each bucket in the index array
// not sure if this is really faster than cpu todo TEST 
// sets helper values
__global__ void set_helper(int *helper, int *code, int *index)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    helper[idx] = code[index[idx]];
}

__global__ void min_helper(int *helper, int *bucket_start, int l, int n_r)
{
    for (int i = threadIdx.x; i < (l * n_r); i += 32)
    {
        
        int v = (helper[i] == blockIdx.x);
        unsigned int mask = __ballot_sync(0xffffffff, v);

        if (__popc(mask) == 0)
            continue;
        int x = __ffs(mask);
        if (threadIdx.x == 0)
        {
            int g = i + x - 1;
            bucket_start[blockIdx.x] = g ; 
            break;
        }
    }
}

// dot two arrays of vectors and set bit corresponding to the dot product  

__global__ void dot_set_bit(float * rand, float * points, int * buckets, int size, int nbits, float * test)
{
   // hipblasHandle_t handle;
   // float res = 0.f ; 
   // hipblasCreate(&handle) ; 
   // hipblasSdot(handle, size, (rand + (size * nbits * threadIdx.x)), 1, (points + (blockIdx.x * size)), 1, &res ) ; 
   //   
   // int bucket = 0 ;
   // if(res >= 0)
   // {    
   //     bucket |= 1UL << threadIdx.y ; 
   // } 
   //  // l n_r nbits 
   // bucket += __shfl_down_sync( 0xffffffff, bucket, 16 );
   // bucket += __shfl_down_sync( 0xffffffff, bucket, 8 ); 
   // bucket += __shfl_down_sync( 0xffffffff, bucket, 4 ); 
   // bucket += __shfl_down_sync( 0xffffffff, bucket, 2 );
   // bucket += __shfl_down_sync( 0xffffffff, bucket, 1 );   
   // if(threadIdx.y == 0)
   // {
   //     //test[] = bucket ;  
   // }
   // //code[ii + i * n_r] |= 1UL << iii;
   // // block = n_r, 1 1
   // // thread = l, 32 , 1 
    
}

// initialize array to a value
__global__ void initialize(int *array, int value)
{

}

// may have to make vectors more random ! hmm todo
// makes a random float

__device__ inline float random_float(uint64_t seed, int idx)
{
    hiprandState s;
    hiprand_init(seed + idx, 0, 0, &s);
    return hiprand_uniform(&s);
}
//fills a vector with random floats
__global__ void random_vector(uint64_t seed, des_t *vec)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float *vector = (float *)vec;
    vector[idx] = random_float(seed, idx);
}