#include "hip/hip_runtime.h"
#include <iostream>
#include "hiprand/hiprand_kernel.h"
#include <string>
#include "hipblas.h"
#include "lsh.h"
#include "knn_brute.h"
#include "helper.h"
#include <algorithm>

// thrust 
#include <thrust/reduce.h>
#include <thrust/gather.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/fill.h>
// use to change rand vectors 
int rand_vec_to_zero  = 0 ; 

void make_vec(int dim, des_t_f &vec)
{
    float * vector = vec ; 
    for (size_t i = 0; i < dim; i++)
    {
     //   if(i == rand_vec_to_zero)
     //   {

     //   vector[i] =  0 ;
     //   }
     //   else{

     //   vector[i] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) -0.5 ;
     //   }
        vector[i] = (static_cast<float>(rand()) / static_cast<float>(RAND_MAX)) -0.5 ;
    } 

    rand_vec_to_zero += 2   ; 
}

class IndexCompare
{
    thrust::counting_iterator<int> _index_copy;
    int* _code ;

public:
    IndexCompare( thrust::counting_iterator<int> index_copy, int* code)
        : _index_copy( index_copy)
        , _code( code)
    { }

    __host__ __device__
    inline bool operator()( int left, int right ) const
    {
        return (_code[_index_copy[left]] < _code[_index_copy[right]]); 
    }
};

void test_alloc(const char * s, hipError_t stat)
{        
    if (stat != hipSuccess) 
    {
        printf ("%s device memory allocation failed \n");
        exit(EXIT_FAILURE) ; 
    }
}

void test_kernel(const char *  s, hipError_t stat)
{
    if (stat != hipSuccess) 
    {
        printf ("%s kernel failed \n");
        exit(EXIT_FAILURE) ; 
    }
}

// n_r, l, 1 
// nbits ,1 ,1 for now we use 32 threads but could also use 16 8 4 2 or 64 / 128  
// set the bit for the one random array then combine whitin warp to form 32 bit buckets 
// want buckets to have layaout 
// bucket[0][0] ......... bucket[0][l]
// bucket[n][0] --------- bucket[n][l]
//
__global__ void set_bit(int *buckets, int nbits, float * dot)
{
    uint32_t var = 0 ; 
    // index of the dot prouduct we need 
    //int dot_idx =  blockIdx.x * gridDim.y * blockDim.x  + blockIdx.y * blockDim.x + threadIdx.x ;    
    int dot_idx =  blockIdx.x * nbits  + threadIdx.x ;    
    // only care if its a relevent thread 
    if((threadIdx.x ) < nbits )
    {
        if((dot[dot_idx] )  >= 0 ) 
        {
            var |= 1UL << threadIdx.x;
        }
    }
    var += __shfl_down_sync( 0xffffffff, var, 16 );
    var += __shfl_down_sync( 0xffffffff, var, 8 ); 
    var += __shfl_down_sync( 0xffffffff, var, 4 ); 
    var += __shfl_down_sync( 0xffffffff, var, 2 );
    var += __shfl_down_sync( 0xffffffff, var, 1 );   
    if(threadIdx.x == 0)
    {
       //buckets[blockIdx.x * gridDim.y + blockIdx.y ] = var ;     
       buckets[blockIdx.x] = var ;     
    }  
}


// not in use using cublas instead  

//want dot array to be 
// point 0 * rand [0][0 - nbits], .......   point n * rand [n][0 - nbits]
// point n * rand [0][0 - nbits], .......   point n * rand [n][0 - nbits] 
// 
// want rand[n] to only be read by blocks on the same sm. load in shared memory  
//  or want points[n ] to only be read by blocks on the same sm TODO  
__global__ void dot_gpu(des_t_f *  rand, des_t_f * points, float *dot)
{
    // called with 
    // n, nbits, l grid  
    //32 1 1 block 
    // could change to 32, x ,1 block todo test if faster 
    
    float res = 0.f ; 
    float4 a = ((float4 * )points[blockIdx.x])[threadIdx.x ];
    // row major 
    //float4 b = ((float4 * )rand[blockIdx.z * gridDim.y + blockIdx.y])[threadIdx.x]; 
    float4 b ; 

    // colom major  
    b.x =  ((float *) rand) [(threadIdx.x * 4) * gridDim.y + blockIdx.y]; 
    b.y =  ((float *) rand) [((threadIdx.x * 4) + 1) * gridDim.y + blockIdx.y]; 
    b.z =  ((float *) rand) [((threadIdx.x * 4) + 2) * gridDim.y + blockIdx.y]; 
    b.w =  ((float *) rand) [((threadIdx.x * 4) + 3) * gridDim.y + blockIdx.y]; 
    res =
        (a.x )*(b.x ) + (a.y )*(b.y ) +
        (a.z )*(b.z ) + (a.w )*(b.w )  ;  
    reduce(res) ; 
    if(threadIdx.x == 0)
    {
        dot[blockIdx.x * gridDim.y * gridDim.z + blockIdx.z * gridDim.y + blockIdx.y] = res ;     
    }  
}

// reduce float 
__device__ inline void reduce(float &var)
{
    var += __shfl_down_sync( 0xffffffff, var, 16 );
    var += __shfl_down_sync( 0xffffffff, var, 8 ); 
    var += __shfl_down_sync( 0xffffffff, var, 4 ); 
    var += __shfl_down_sync( 0xffffffff, var, 2 );
    var += __shfl_down_sync( 0xffffffff, var, 1 );   
}


// not usable atm
// called with 
// block, nbits, x, 1 x = is what ever number we need to make 3 warps nbits = 32 -> 3 31 -> 3 16 -> 6 8-> 12       
// grid, (bucket_n * 2 /number of sm), 1, 1.  
//
// want to maximize use of shared memory so there is max one read from global memory per bucket 
//    
__global__ void find_all_neigbours_dist_1(int to_read, int * neighbouring_buckets, int nbits, int * bucket, int n_buckets ) 
{
    // read all the buckets from global memory
    // read n per sm / block 
    __shared__ int buckets ;
    if(threadIdx.x == 0)
    {
        buckets = bucket[blockIdx.x] ; 
    }
    __syncthreads() ; 

    int neigbour = buckets ; 
    neigbour ^= 1UL << threadIdx.x ; 
    neighbouring_buckets[threadIdx.x + n_buckets * blockIdx.x] = neigbour ; 
}

__global__ void find_all_neigbours_dist_2_odd(int to_read, int * neighbouring_buckets, int nbits, int * bucket ) 
{
     // read all the buckets from global memory
    // read n per sm / block 
    __shared__ int buckets ;
    if(threadIdx.x == 0 && threadIdx.y == 0)
    {
        buckets = bucket[blockIdx.x] ; 
    }
    __syncthreads() ; 

    int neigbour = buckets ; 
    neigbour  ^= 1UL << threadIdx.x ; 
    neigbour  ^= 1UL << ((threadIdx.x + 1 + threadIdx.y) % nbits)  ; 

    neighbouring_buckets[blockIdx.x * (blockDim.x * blockDim.y + nbits) + nbits + threadIdx.x * blockDim.y + threadIdx.y ] = neigbour ; 
}
__global__ void find_all_neigbours_dist_2_pair(int to_read, int * neighbouring_buckets, int nbits, int * bucket ) 
{
    // read all the buckets from global memory
    // read n per sm / block 
    __shared__ int buckets ;
    if(threadIdx.x == 0 && threadIdx.y == 0)
    {
        buckets = bucket[blockIdx.x] ; 
    }
    __syncthreads() ; 

    int neigbour = buckets ; 
    int val = 0  ; 
    if(threadIdx.y + 1 == blockDim.y )
    {
        val = blockDim.x ; 
    }
    else
    {
        val = ((threadIdx.x + 1 + threadIdx.y) % nbits) ;  
    }
    neigbour  ^= 1UL << threadIdx.x ; 
    neigbour  ^= 1UL << val; 

    neighbouring_buckets[blockIdx.x * (blockDim.x * blockDim.y + nbits) + nbits + threadIdx.x * blockDim.y + threadIdx.y ] = neigbour ; 
}

//find smallest vlaue in the warp and index  
__device__ inline void best_in_warp(float4  &min_2)
{    
    for (int i = 16; i > 0; i/= 2)
    {          
        float x_dist = __shfl_down_sync( 0xffffffff, min_2.x, i );
        float y_dist = __shfl_down_sync( 0xffffffff, min_2.y, i );
        float w_value = __shfl_down_sync( 0xffffffff, min_2.w, i );
        float z_value = __shfl_down_sync( 0xffffffff, min_2.z, i );
        if(x_dist < min_2.x)
        {
            min_2.y = min_2.x ; 
            min_2.x = x_dist ;  
                
            min_2.w = min_2.z ; 
            min_2.z = z_value;  
        }
        else{
            if(x_dist < min_2.y)
            {
                min_2.y = x_dist ; 
                min_2.w = z_value;  
                continue ; 
            }
        } 
        if(y_dist < min_2.y)
        {
                min_2.y = y_dist ; 
                min_2.w = w_value;  
        }
    }
}

__device__ inline float4 set_sorted(float4 sorted , float4 min)
{
    if(sorted.x > min.x)
    {
        if(sorted.x > min.y)
        {
            sorted.y = min.y ; 
            sorted.x = min.x ; 
            sorted.w = min.w ;  
            sorted.z = min.z ;  
        }
        else
        {
            sorted.y = sorted.x ; 
            sorted.w = sorted.z ; 
            sorted.x = min.x ; 
            sorted.z = min.z ; 
        }
    }
    else
    {
        if (sorted.y > min.x)
        {
            sorted.y = min.x ; 
            sorted.w = min.z ; 
        }
    }
    return sorted ; 
}
// takes 2 buckets and find the 2nns 
// called with 
// grid, y, 1, 1 
// block 32, x, 1 
__global__ void brute_2nn(float4 * sorted, int * index_r, int * index_q, int4 * start_size, des_t_f * r_p, des_t_f *  q_p) 
{
    // use a int for now to test 
    int r_size = 4 ; 
    int4 start_size_q_r = start_size[blockIdx.x] ; 

    __shared__ float4 r_points[32 * 4]   ;
    // dose not need to be shared hmm 
    float4 best ; 
     
    // for each q point 
    for (int i = 0; i < start_size_q_r.y; i += r_size) 
    {
        float4 a ; 
        int count = 0 ; 
        // set shared value and read in q point 
        if((i + threadIdx.y) < start_size_q_r.y)
        {
            // read new q point 
            a = ((float4 * )q_p[index_q[start_size_q_r.w + (threadIdx.y + i)]])[threadIdx.x]; 
            best.x = MAXFLOAT ; 
            best.y = MAXFLOAT ; 
        } 
        // for every r point find dist to q points we have read in  
        for (int ii = 0; ii < start_size_q_r.z ; ii += r_size)
        {
            // read to shared ? 
            __syncthreads(); 
            if((ii + threadIdx.y) < start_size_q_r.z)
            {
                r_points[32 * threadIdx.y + threadIdx.x] = ((float4 * )r_p[index_r[start_size_q_r.x + (threadIdx.y + ii)]])[threadIdx.x];  
                // add if stamtemnt and cal dist for this here ?_? maybe
            }
            __syncthreads() ; 
            if((i + threadIdx.y) < start_size_q_r.y)
            {
                
                int iii = 0 ; 
                while (((iii + ii) < start_size_q_r.z ) && iii < 4)
                {
                    
                    float res = 0.f ; 
                    float4 b = r_points[threadIdx.x + iii * 32]  ; 
                    float4 c ; 
                    c.x = a.x - b.x ; 
                    c.y = a.y - b.y ; 
                    c.z = a.z - b.z ; 
                    c.w = a.w - b.w ; 

                    res =
                    (c.x )*(c.x ) + (c.y )*(c.y ) +
                    (c.z )*(c.z ) + (c.w )*(c.w )  ;  
                    reduce(res) ;    
                    res = __shfl_sync(0xFFFFFFFF, res, 0 ) ;

                    // set value 
                    if(threadIdx.x == count)
                    {
                        if(best.x == MAXFLOAT)
                        {
                            best.x = res ;  
                            best.z = index_r[iii + start_size_q_r.x + ii] ; 
                        }
                        // will never be reached atm 
                        else
                        {
                            best.y = res ;  
                            best.w = index_r[iii + start_size_q_r.x + ii] ; 
                        }
                    } 

                    iii ++ ; 
                    count +=1 ;  

                    if(count == 32)
                    {
                        best_in_warp(best) ; 
                        count = 0 ; 
                        if(threadIdx.x == 0)
                        {
                            // could also keep the valus in shared so there is no need to read from sorted more than once hmmm 
                            sorted[index_q[start_size_q_r.w + (threadIdx.y + i)]] = set_sorted(sorted[index_q[start_size_q_r.w + (threadIdx.y + i)]], best ); 
                        }
                        best.x = MAXFLOAT ;
                        best.y = MAXFLOAT ;  
                    }
                }
            }
            __syncthreads() ; 
        }

        if((i + threadIdx.y) < start_size_q_r.y)
        {
            best_in_warp(best) ; 
            count = 0 ; 
            if(threadIdx.x == 0)
            {
                sorted[index_q[start_size_q_r.w + (threadIdx.y + i)]] = set_sorted(sorted[index_q[start_size_q_r.w + (threadIdx.y + i)]], best ); 
            }
            best.x = MAXFLOAT ;
            best.y = MAXFLOAT ;  
        }
    }   
}



void lsh_test(des_t_f *q_points, des_t_f *r_points, int n_q, int n_r, float4 *sorted, int nbits, int l, int max_dist, hipblasHandle_t handle) 
{  
    // see how much memory we have  
    size_t free_byte ;
    size_t total_byte ;
    hipMemGetInfo( &free_byte, &total_byte ) ;

    double free_db = (double)free_byte ;
    double total_db = (double)total_byte ;
    double used_db = total_db - free_db ;

    printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

         used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
    

    int size_bucket = 0 ;
    if(max_dist == 1)
    {
        size_bucket = nbits ; 
    }
    else{
        size_bucket = ((nbits * (nbits -1 )) / 2) + nbits ;  
    }

    // not accurate atm
    //printf("we need %i mb of space ",(((n_r * 4 * 4)+ (n_q * 4 * 4) + sizeof(int) * n_q * size_bucket) + nbits * 4 * n_q + nbits * 4 * n_r + 4 * 128 * nbits) / 1024 ) ; 

    // arry of vectors 
    des_t_f *rand_array;

    // hash codes  
    int *code_r, *code_q;

    // index into bucket array and copy to sort 
    int *index_r, * index_q; 

    // all buckets in use 
    int *buckets_r, *buckets_q; 

    // size of each of the buckets 
    int * buckets_r_size, * buckets_q_size ;  

    // used to reduce by key 
    int * code_by_index_r, * code_by_index_q ; 
    // will give us index_copy[0 -> N] = 0 -> N  
    // used to index into the buckets 
    thrust::counting_iterator<int> index_copy(0);
    
    // will always give us 1 
    // is used to both find number of elemets in each bucket and make an array of all in use buckets 
    thrust::constant_iterator<int> array_of_ones(1) ; 

    // dot from random vector to q / r points 
    float * dot_res_r, * dot_res_q;
 
    // a bucket of all the points each q has to check   
    int *neighbouring_buckets;
    // number of buckets within hamming distance r given n bits
    hipMallocManaged((void **)&neighbouring_buckets, sizeof(int) * n_q * size_bucket);

    hipMallocManaged((void **)&rand_array, sizeof(des_t_f) * nbits);

    hipMallocManaged((void **)&index_r, sizeof(int) * n_r);  
    hipMallocManaged((void **)&index_q, sizeof(int) * n_q);  
 
    // need to index with a smaller array 
    // this would in the worst case use 17 gb of memory :(
    // hipMallocManaged((void **)&bucket_start_r, (2 << nbits) * sizeof(int));

    hipMallocManaged((void **)&code_r, sizeof(int) * n_r);
    hipMallocManaged((void **)&code_q, sizeof(int) * n_q);

    hipMallocManaged((void **)&dot_res_r, nbits * n_r* sizeof(float)); 
    hipMallocManaged((void **)&dot_res_q, nbits * n_q* sizeof(float));

    hipMallocManaged((void **)&buckets_r_size, sizeof(int) * n_r);
    hipMallocManaged((void **)&buckets_r, sizeof(int) * n_r);
    hipMallocManaged((void **)&code_by_index_r, sizeof(int) * n_r);
    
    hipMallocManaged((void **)&buckets_q_size, sizeof(int) * n_q);
    hipMallocManaged((void **)&buckets_q, sizeof(int) * n_q);
    hipMallocManaged((void **)&code_by_index_q, sizeof(int) * n_q);

    //fill sorted with MAXFLOAT 
    thrust::fill(thrust::device,(float * )sorted,(float*)( sorted + n_q * 4), MAXFLOAT) ; 

    // cublas 
    float a = 1.0f;
    float b = 0.0f;
    
    IndexCompare code_r_sort(index_copy, code_r);
    IndexCompare code_q_sort(index_copy, code_q);
    // thrust pointers for q 
    // todo check if we need thrust pointers 
    thrust::device_ptr<int> ptr_q_index = thrust::device_pointer_cast(index_q);
    thrust::device_ptr<int> ptr_code_by_index_q = thrust::device_pointer_cast(code_by_index_q);
    thrust::device_ptr<int> ptr_code_q = thrust::device_pointer_cast(code_q);
    thrust::device_ptr<int> ptr_buckets_q = thrust::device_pointer_cast(buckets_q);
    thrust::device_ptr<int> ptr_buckets_q_size = thrust::device_pointer_cast(buckets_q_size);
    // thrust pointers for r
    thrust::device_ptr<int> ptr_r_index = thrust::device_pointer_cast(index_r);
    thrust::device_ptr<int> ptr_code_by_index_r = thrust::device_pointer_cast(code_by_index_r);
    thrust::device_ptr<int> ptr_code_r = thrust::device_pointer_cast(code_r);
    thrust::device_ptr<int> ptr_buckets_r = thrust::device_pointer_cast(buckets_r);
    thrust::device_ptr<int> ptr_buckets_r_size = thrust::device_pointer_cast(buckets_r_size);
    for (int L = 0; L < l; L++)
    {
        // memsetstuff
        hipMemset(neighbouring_buckets, 0, sizeof(int) * n_q * size_bucket);

        // set index arrays  
        thrust::copy(index_copy,index_copy+ n_q,index_q) ;  
        thrust::copy(index_copy,index_copy+ n_r,index_r) ;  

        // to do random vectos gpu hiprand / thrust 
       // make random vectors
        for (int i = 0; i < nbits; i++)
        {
            make_vec(128, rand_array[i]);
        }
        //using to see how setting values in the rand vec to 0 changes things         
        rand_vec_to_zero = l ; 


        // dot random vectors with n_r
        // using cublas
        //dim3 grid_dot_r(n_r, nbits, 1) ;
        //dim3 block_dot_r(32, 1, 1) ;   
        //dot_gpu<<<grid_dot_r, block_dot_r>>>(rand_array, r_points, dot_res_r); 

        //cublas dot
        // note the rand array is read as colum major not row major 
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nbits, n_r, 128, &a, (float *)rand_array, nbits, (float *)r_points, 128, &b, dot_res_r, nbits);

        
        // set bit for code_r 
        dim3 grid_bit_r(n_r,1,1) ; 
        dim3 block_bit_r(32,1,1) ; 
        set_bit<<<grid_bit_r, block_bit_r>>>(code_r, nbits, dot_res_r) ; 

        // dot random vectors with q
        //dim3 grid_dot_q(n_q, nbits, 1) ;
        //dim3 block_dot_q(32, 1, 1) ;   
        //dot_gpu<<<grid_dot_q, block_dot_q>>>(rand_array, q_points, dot_res_q); 

        //cublas dot
        // note the rand array is read as colum major not row major 
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, nbits, n_q, 128, &a, (float *)rand_array, nbits, (float *)q_points, 128, &b, dot_res_q, nbits);

        // set bit for hash values for code_q 
        dim3 grid_bit_q(n_r,1,1) ; 
        dim3 block_bit_q(32,1,1) ; 
        set_bit<<<grid_bit_q, block_bit_q>>>(code_q, nbits, dot_res_q) ; 


        // sort and reduce for r buckets  
        thrust::sort(ptr_r_index, ptr_r_index + n_r, code_r_sort );
        thrust::gather(thrust::device, ptr_r_index, ptr_r_index + n_r, ptr_code_r, ptr_code_by_index_r) ; 
        auto new_end_r = thrust::reduce_by_key( ptr_code_by_index_r, ptr_code_by_index_r+ n_r, array_of_ones, ptr_buckets_r, ptr_buckets_r_size) ; 

        // sort and reduce for q buckets 
        thrust::sort(ptr_q_index, ptr_q_index + n_q, code_q_sort );
        thrust::gather(thrust::device, ptr_q_index, ptr_q_index + n_q, ptr_code_q, ptr_code_by_index_q) ; 
        auto new_end_q = thrust::reduce_by_key( ptr_code_by_index_q, ptr_code_by_index_q+ n_q, array_of_ones, ptr_buckets_q, ptr_buckets_q_size) ; 

        int n_r_buckets = new_end_r.first - (ptr_buckets_r) ; 
        int n_q_buckets = new_end_q.first - (ptr_buckets_q) ; 

        // todo fix dist only need dist one  
       // if(max_dist > 0){
       //      
       //     dim3 grid_bucket(n_q_buckets, 1, 1) ; 
       //     dim3 block_bucket(nbits, 1 ,1) ;                
       //     find_all_neigbours_dist_1<<<grid_bucket, block_bucket>>>(1, neighbouring_buckets, nbits, buckets_q, size_bucket) ; 

       //     if(max_dist == 2) 
       //     {
       //         if(nbits % 2)
       //         {
       //             dim3 grid_bucket(n_q, 1, 1) ; 
       //             dim3 block_bucket(nbits,(nbits - 1) / 2 ,1) ;                
       //             find_all_neigbours_dist_2_odd<<<grid_bucket, block_bucket>>>(1, neighbouring_buckets, nbits, code_q) ; 
       //         }
       //         else
       //         {
       //             dim3 grid_bucket(n_q, 1, 1) ; 
       //             dim3 block_bucket(nbits - 1,(nbits) / 2 ,1) ;                
       //             find_all_neigbours_dist_2_pair<<<grid_bucket, block_bucket>>>(1, neighbouring_buckets, nbits, code_q) ; 
       //         }
       //     }
       // }

        int count_r = 0 ;  
        int count_q = 0 ;  

        int start_index_r = 0 ; 
        int start_index_q = 0 ; 
        
        // what each block in our kernel needs is 
        // number of r and q points to read 
        // start index 
        // index of each q point into the soreted array 

        // we can use block number to index into the array
        int4 * index_size_start ; 
        hipMallocManaged((void **) &index_size_start, sizeof(int) * n_q_buckets) ; 
        int counter = 0 ;  
        // can this be done on gpu ? threads ? todo 
        while (count_q < n_q_buckets && count_r < n_r_buckets)
        {
            if(buckets_q[count_q] == buckets_r[count_r])
            {
                index_size_start[counter].w = start_index_q ; 
                index_size_start[counter].x = start_index_r ; 
                index_size_start[counter].y = buckets_q_size[count_q] ; 
                index_size_start[counter].z = buckets_r_size[count_r] ; 

                // number of points in each bucket 
                //printf("%i == %i bucket r size = %i bucket q size = %i  \n" , buckets_r[count_r], buckets_q[count_q], buckets_r_size[count_r],buckets_q_size[count_q]) ; 

                start_index_q += buckets_q_size[count_q]; 
                start_index_r += buckets_r_size[count_r]; 
                count_r ++ ; 
                count_q ++ ; 
                counter ++ ; 
            }
            else if( buckets_q[count_q] < buckets_r[count_r])
            {
                start_index_q += buckets_q_size[count_q]; 
                count_q ++ ; 
            }
            else
            {
                start_index_r += buckets_r_size[count_r]; 
                count_r ++ ; 
            }
        }
           
        dim3 brute_grid(counter, 1, 1)  ;  
        dim3 brute_bucket(32, 4, 1) ; 
        brute_2nn<<<brute_grid, brute_bucket>>>(sorted, index_r, index_q, index_size_start ,r_points, q_points) ; 
        hipFree(index_size_start) ; 
   } 
    hipFree(neighbouring_buckets) ; 
    hipFree(rand_array) ; 

    hipFree(index_q) ; 
    hipFree(index_r) ; 

    hipFree(code_q) ; 
    hipFree(code_r) ; 

    hipFree(dot_res_q) ; 
    hipFree(dot_res_r) ; 

    hipFree(buckets_q_size) ; 
    hipFree(code_by_index_q) ; 
    hipFree(buckets_r) ; 
    hipFree(buckets_q) ; 

    hipFree(buckets_r_size) ; 
    hipFree(code_by_index_r) ; 
}


