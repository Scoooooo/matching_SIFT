#include "hip/hip_runtime.h"
#include <iostream>
#include "hiprand/hiprand_kernel.h"
#include <string>
#include "hipblas.h"
#include "lsh.h"
#include "knn_brute.h"
#include <memory>
#include "helper.h"
#include <vector>
#include <algorithm>
#include <bits/stdc++.h>

#define PREFER_CPU 0

#if PREFER_CPU == 0
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#endif

using namespace std;
int fact(int n){
    return (n==1 || n==0) ? 1: n * fact(n - 1);
}
void host_lsh(des_t *q_points, des_t *r_points, int n_q, int n_r, float4 *sorted, int nbits, int l, int max_dist)
{

    des_t *rand_array;
    hipMallocManaged((void **)&rand_array, sizeof(des_t) * nbits * l);
    // make random vectors
    for (size_t i = 0; i < l * nbits; i++)
    {
        make_vec(128, rand_array[i]);
    }
    // make an array of ints with one int for each r_point
    int *code;
    int * code_test ; 
    int *index;
    // need a copy to sort using sort
    int *index_copy;
    int *bucket_start;

    int * index_test ; 
    int * index_copy_test ; 
    hipMallocManaged((void **)&index, sizeof(int) * n_r * l);
    hipMemset(index, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&index_copy, sizeof(int) * n_r * l);
    hipMemset(index_copy, 0, sizeof(int) * n_r * l);
    
    hipMallocManaged((void **)&index_test, sizeof(int) * n_r * l);
    hipMemset(index_test, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&index_copy_test, sizeof(int) * n_r * l);
    hipMemset(index_copy_test, 0, sizeof(int) * n_r * l);


    hipMallocManaged((void **)&bucket_start, (2 << nbits) * sizeof(int));

    for (int i = 0; i < (2 << nbits); i++)
    {
        bucket_start[i] = -1;
    }

    hipMallocManaged((void **)&code, sizeof(int) * n_r * l);
    hipMemset(code, 0, sizeof(int) * n_r * l);
    hipMallocManaged((void **)&code_test, sizeof(int) * n_r * l);
    hipMemset(code_test, 0, sizeof(int) * n_r * l);


    // test if our gpu dot works 
    float * dot_res;
    hipMallocManaged((void **)&dot_res, l * nbits * n_r* sizeof(float));
    double s = start_timer() ; 
    dim3 grid(n_r, nbits, l) ;
    dim3 block(32, 1, 1) ;   
    dot_gpu<<<grid, block>>>(rand_array, r_points, dot_res); 
    hipDeviceSynchronize();
    grid.z = 1 ; 
    grid.y = l ; 
    block.x = nbits ;
    set_bit<<<grid, block>>>(code_test, nbits, dot_res) ; 
    hipDeviceSynchronize();
    print_time(s, "gpu") ; 
    s = start_timer() ; 
    for (int i = 0; i < n_r; i++)
    {
        for (int ii = 0; ii < l; ii++)
        {
            for (int iii = 0; iii < nbits ; iii++)
            {
                float sum = dot(r_points[i] ,rand_array[iii + ii * nbits] );
          //     printf("sum = %f ", sum) ; 
          //     printf("gpu = %f \n", dot_res[i * nbits * l + ii * nbits + iii]) ; 
                if (sum <= 0)
                {
                    code[i * l + ii ] |= 1UL << iii;
                }

            }
                if(code[i * l + ii] != code_test[i * l + ii])
                {
                    printf("cpu = %i ", code[i * l + ii]) ; 
                    printf("gpu = %i \n", code_test[i*l + ii] ) ;                       
 
                }
       }
    }
    print_time(s, "cpu") ; 
     
    
 //   // dot all vectors and add the bit to the coresponding int bit for the r points
 //   for (int i = 0; i < l; i++)
 //   {
 //       for (int ii = 0; ii < n_r; ii++)
 //       {
 //           //            printf(" %i bucket = ", (ii + i * n_r));
 //           for (int iii = 0; iii < nbits; iii++)
 //           {
 //               float sum = dot(r_points[ii], rand_array[iii + i * nbits]);
 //               if (sum <= 0)
 //               {
 //                   code[ii + i * n_r] |= 1UL << iii;
 //               }
 //           }
 //           //           printf(" %i ", code[ii + i * n_r]);
 //           //           printf("\n \n");
 //       }
 //   }

    // make buckets for r points
    grid.x = n_r/32 +1 ; 
    grid.y = l ; 
    block.x = 32 ; 
    block.y = 3 ; 
    set_bucket<<<grid,block>>>(index_test, index_copy_test, n_r) ; 
    hipDeviceSynchronize();
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_r; ii++)
        {
            index[ii + i * n_r] = ii;
            index_copy[ii + i * n_r] = ii;
        }
    }


    std::sort(index, index + n_r * l, [&](const int &i, const int &j) -> bool
              { return (code[index_copy[i]] < code[index_copy[j]]); });
//    std::sort(index_test, index_test + n_r * l, [&](const int &i, const int &j) -> bool
//              { return (code_test[index_copy_test[i]] < code_test[index_copy_test[j]]); });


   // s = start_timer(); 
   // int *helper;
   // hipMallocManaged((void **)&helper, sizeof(int) * n_r * l);

   // //set bucket start
   // dim3 block_size(32, 1, 1);
   // dim3 grid_size(((l * n_r) / 32) + 1, 1, 1);
   // set_helper<<<grid_size, block_size>>>(helper, code, index);

   // hipDeviceSynchronize();
   // dim3 blocks(32, 1, 1);
   // dim3 grids(n_r * l, 1, 1);

   // set_bucket_start<<<grids, blocks>>>(helper, bucket_start, l, n_r);
   // hipDeviceSynchronize();
   // print_time(s,"set start gpu");

    s = start_timer(); 
    for (int i = 0; i < n_r * l; i++)
    {
        if (bucket_start[code[index[i]]] == -1)
        {
            bucket_start[code[index[i]]] = i;
        }
    }
    print_time(s,"set start cpu");

    int *code_q;
    hipMallocManaged((void **)&code_q, sizeof(int) * n_r * l);
    hipMemset(code_q, 0, sizeof(int) * n_r * l);
    // dot all q points
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_q; ii++)
        {
            for (int iii = 0; iii < nbits; iii++)
            {
                float sum = dot(q_points[ii], rand_array[iii + i * nbits]);
                if (sum <= 0)
                {
                    code_q[ii + i * n_q] |= 1UL << iii;
                }
            }
        }
    }

    int *buckets;
    hipMallocManaged((void **)&buckets, sizeof(int) * n_q * n_r);
    hipMemset(buckets, 0, sizeof(int) * n_q * n_r);
    // fill buckets
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_q; ii++)
        {
            int bucket = code_q[ii + (i * n_q)];
            int start = bucket_start[bucket];
            int iii = start;
            while ((start != -1) && code[index[iii]] == bucket)
            {
                buckets[ii * n_r + index[iii]] = 1;
                iii++;
            }

            // add from negbouring buckets up to n bits away // n can be given by input but is by defualt 1
            // is there any meaing to adding more than

            // make all buckets with a hamming distance of n

            // 0000
            // 1000 0100 0010 0001
            // 1100 1010 1001
            // 0110 0101
            // 0011

            // 010
            // 011 000 110
            // 001 100
            for (int n = 0; n < max_dist; n++)
            {
                int counters[n + 1];

                for (int q = 0; q < (n + 1); q++)
                {
                    counters[q] = q;
                }

                bool done = false;
                while (!done)
                {
                    int neighbour_bucket = bucket;
                    for (int nn = 0; nn < (n + 1); nn++)
                    {
                        neighbour_bucket ^= 1UL << counters[nn];
                    }
                    // printf("bucket is %i neighbour is %i \n", bucket, neighbour_bucket) ;
                    // we have bucket
                    int start = bucket_start[neighbour_bucket];
                    int iii = start;
                    while ((start != -1) && code[index[iii]] == neighbour_bucket)
                    {
                        buckets[ii * n_r + index[iii]] = 1;
                        iii++;
                    }
                    bool flag = false;
                    int nnn = n;
                    int bits = nbits;
                    while (!flag)
                    {

                        if (((counters[nnn] + 1) >= bits) && nnn == 0)
                        {
                            flag = true;
                            done = true;
                        }
                        else if ((counters[nnn] + 1) < bits)
                        {
                            counters[nnn] += 1;
                            flag = true;
                        }

                        nnn--;
                        bits--;
                    }
                }
            }
            //         printf("\n");
        }
    }

    for (int i = 0; i < n_q; i++)
    {
        sorted[i].w = MAXFLOAT;
        sorted[i].x = MAXFLOAT;
        sorted[i].y = MAXFLOAT;
        sorted[i].z = MAXFLOAT;
        for (int ii = 0; ii < n_r; ii++)
        {
            if (buckets[n_r * i + ii] == 1)
            {
                float dist = host_lenght(r_points[ii], q_points[i]);

                if (dist < sorted[i].x)
                {
                    sorted[i].y = sorted[i].x;
                    sorted[i].x = dist;

                    sorted[i].w = sorted[i].z;
                    sorted[i].z = ii;
                }
                else
                {
                    if (dist < sorted[i].y)
                    {
                        sorted[i].y = dist;
                        sorted[i].w = ii;
                    }
                }
            }
        }
    }
}

void make_vec(int dim, des_t &vec)
{
    for (size_t i = 0; i < dim; i++)
    {
        vec[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }
}

float dot(des_t v1, des_t v2)
{

    float sum = 0.f;
    for (size_t i = 0; i < 128; i++)
    {
        float a = ((float *)v1)[i] -0.5;
        float b = ((float *)v2)[i]- 0.5;
        float c = a * b;
        sum += c;
    }
    return sum;
}
float test_dot(float *v1, des_t v2)
{
    float sum = 0.f;
    for (size_t i = 0; i < 128; i++)
    {
        float a = ((float *)v1)[(4 * 4) * i];
        float b = ((float *)v2)[i];
        float c = a * b;
        sum += c;
    }
    return sum;

}

class IndexCompare
{
    int* _index_copy;
    int* _code ;

public:
    IndexCompare( int* index_copy, int* code)
        : _index_copy( index_copy)
        , _code( code)
    { }

    __host__ __device__
    inline bool operator()( int left, int right ) const
    {
        return (_code[_index_copy[left]] < _code[_index_copy[right]]); 
    }
};

void sort_bucket( )
{

}

//void sort_buckets()

// kernels for finding the start of each bucket in the index array
// not sure if this is really faster than cpu todo TEST
// sets helper values
__global__ void set_helper(int *helper, int *code, int *index)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    helper[idx] = code[index[idx]];
}

__global__ void set_bucket_start(int *helper, int *bucket_start, int l, int n_r)
{
    for (int i = threadIdx.x; i < (l * n_r); i += 32)
    {

        int v = (helper[i] == blockIdx.x);
        unsigned int mask = __ballot_sync(0xffffffff, v);

        if (__popc(mask) == 0)
            continue;
        int x = __ffs(mask);
        if (threadIdx.x == 0)
        {
            int g = i + x - 1;
            bucket_start[blockIdx.x] = g;
            break;
        }
    }
}
// n_r, l, 1 
// nbits ,1 ,1 for now we use 32 threads but could also use 16 8 4 2 or 64 / 128  
// set the bit for the one random array then combine whitin warp to form 32 bit buckets 
// want buckets to have layaout 
// bucket[0][0] ......... bucket[0][l]
// bucket[n][0] --------- bucket[n][l]
//
__global__ void set_bit(int *buckets, int nbits, float * dot)
{
    int var = 0 ; 
    // index of the dot prouduct we need 
    
    int dot_idx =  blockIdx.x * gridDim.y * blockDim.x  + blockIdx.y * blockDim.x + threadIdx.x ;    
    // only care if its a relevent thread 
    if((threadIdx.x ) < nbits )
    {
        if(dot[dot_idx] <= 0 ) 
        {
            var |= 1UL << threadIdx.x;
        }
    }
    var += __shfl_down_sync( 0xffffffff, var, 16 );
    var += __shfl_down_sync( 0xffffffff, var, 8 ); 
    var += __shfl_down_sync( 0xffffffff, var, 4 ); 
    var += __shfl_down_sync( 0xffffffff, var, 2 );
    var += __shfl_down_sync( 0xffffffff, var, 1 );   
    if(threadIdx.x == 0)
    {
        buckets[blockIdx.x * gridDim.y + blockIdx.y ] = var ;     
    }  
}
//want dot array to be 
// point 0 * rand [0][0 - nbits], .......   point n * rand [n][0 - nbits]
// point n * rand [0][0 - nbits], .......   point n * rand [n][0 - nbits] 
// 
// want rand[n] to only be read by blocks on the same sm. load in shared memory  
//  or want points[n ] to only be read by blocks on the same sm TODO  
__global__ void dot_gpu(des_t *  rand, des_t * points, float *dot)
{
    // called with 
    // n, nbits, l grid  
    //32 1 1 block 
    // could change to 32, x ,1 block todo test if faster 0_0 
    
    float res = 0.f ; 
    float4 a = ((float4 * )points[blockIdx.x])[threadIdx.x ];
    float4 b = ((float4 * )rand[blockIdx.z * gridDim.y + blockIdx.y])[threadIdx.x]; 

    res +=
        (a.x -0.5)*(b.x - 0.5) + (a.y -0.5)*(b.y -0.5) +
        (a.z -0.5)*(b.z - 0.5) + (a.w -0.5)*(b.w -0.5)  ;  
    reduce(res) ; 
    if(threadIdx.x == 0)
    {
        dot[blockIdx.x * gridDim.y * gridDim.z + blockIdx.z * gridDim.y + blockIdx.y] = res ;     
    }  
}


// initialize array to a value
__global__ void initialize(int *array, int value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    array[idx] = value;
}

// may have to make vectors more random ! hmm todo
// makes a random float

__device__ inline float random_float(uint64_t seed, int idx)
{
    hiprandState s;
    hiprand_init(seed + idx, 0, 0, &s);
    return hiprand_uniform(&s);
}
//fills a vector with random floats
__global__ void random_vector(uint64_t seed, des_t *array)
{
    int idx = threadIdx.x + threadIdx.y * 32;
    float * vec = (float *)array[blockIdx.x]  ;
    vec[idx] = random_float(seed, idx);
    
}
__device__ inline void reduce(float &var)
{
    var += __shfl_down_sync( 0xffffffff, var, 16 );
    var += __shfl_down_sync( 0xffffffff, var, 8 ); 
    var += __shfl_down_sync( 0xffffffff, var, 4 ); 
    var += __shfl_down_sync( 0xffffffff, var, 2 );
    var += __shfl_down_sync( 0xffffffff, var, 1 );   
}
// called with 
// only works up to a distance of 3 
//
__global__ void find_all_neigbours(int * neighbouring_buckets, int nbits, int dist, int n_q, int * bucket ) 
{

    // could read more codes to use fewer threads ?
    __shared__ int code[] ;  
    if(threadIdx.x == 0)
    {

        code = bucket[blockIdx.x] ; 
    }
    
    // 0000
    // 1000 0
    if(dist == 1)
    {
        int neigbour = code ;   
        neigbour ^= 1UL << threadIdx.x ; 
        neighbouring_buckets[threadIdx.x + n_q * blockIdx.x] = neigbour ; 

        return ; 
    }  

// 5, 4 
// 5, 2 
// 11000 10100 01100 01010 00110 00101 00011 10010 10001 01001
// set(1, 1+1), set(1, 1+2)
// set(2, 2+1), set(2, 2+2)
// set(3, 3+1), set(3, 3+2)
// set(4, 4+1), set(4, 4+2) over 5 rolls over and back to 0 
// set(5, 5+1), set(5, 5+2)

// 7 * 6 / 2 

// nbits = 7 dist = 2 gives us 7*6/2 = 21 buckets sice nbits is odd we can divide nbits - 1 so 6/2. gives us 7 * 3 threads  
//7,3 
// set(1, 1+1), set(1, 1+2) set(1, 1+3) // 1100000 1010000 1001000 
// set(2, 2+1), set(2, 2+2) set(2, 2+3) // 0110000 0101000 0100100
// set(3, 3+1), set(3, 3+2) set(3, 3+3) // 0011000 0010100 0010010 
// set(4, 4+1), set(4, 4+2) set(4, 4+3) // 0001100 0001010 0001001 
// set(5, 5+1), set(5, 5+2) set(5, 5+3) // 0000110 0000101 1000100 
// set(6, 6+1), set(6, 6+2) set(6, 6+3) // 0000011 1000010 0100010 
// set(7, 7+1), set(7, 7+2) set(7, 7+3) // 1000001 0100001 0010001
// works for odd nbits  hm
    if(dist == 2)
    {
        int neigbour = code ; 
        neigbour  ^= 1UL << threadIdx.x ; 
        neigbour  ^= 1UL << ((threadIdx.x + 1 + threadIdx.y) % nbits)  ; 
    }

// 6, 5
// 3, 5
// 5, 3 ? 
// set(1, 1+1), set(1, 1+2), set(1, 1+3), set(1, 1+4), set(1, 1+5) // 110000 101000 010100 010010 010001 
// set(2, 2+1), set(2, 2+2), set(2, 2+3), set(2, 2+4), set(2, 2+5) // 011000 010100 001010 001001 101000 
// set(3, 3+1), set(3, 3+2), set(3, 3+3), set(3, 3+4), set(3, 3+5) //  

// //110000 101000 100100 
// //011000 010100 010010
// //001100 001010 001001
// //000110 000101 100100
// //000011 100010 010010
// pair 
    if(dist == 2)
    {

    }
}

// CALLED WITH 
// grid n_q, 1, 1 
//block max_dist, 1, 1 
__global__ void hamming(int * neighbouring_buckets, int dist, int size, int * bucket )
{
    int start = bucket[blockIdx.x] ; 
    change_bit(size, threadIdx.x, 1, 0, start) ; 
}

 __device__ void change_bit(int n, int k, int dir, int pos, int start)
{
    for (size_t i = 1; i <= n - k + 1; i++, pos += dir)
    {
        start ^= 1UL << pos ; 
        if (k > 1) 
        {
            change_bit(n - i, k - 1, i % 2 ? dir : -dir, pos + dir * (i % 2 ? 1 : n - i), start);
        }
        else 
        {
            printf("int is %i \n", start) ; 
        }
        start ^= 1UL << pos ; 
    }
}
// called with 
// grid n_r/32 +1 l 1         
// block 32 3 1 
__global__ void set_bucket(int * index, int * index_copy, int n)
{
    int i = blockDim.x * blockDim.y * blockIdx.x +  blockDim.x * threadIdx.y + threadIdx.x ; 
    if(n > i)
    {
        index[i + blockIdx.y * n] = i ; 
        index_copy[i + blockIdx.y * n] = i ; 
    }
}
//    for (int i = 0; i < l; i++)
//    {
//        for (int ii = 0; ii < n_r; ii++)
//        {
//            index[ii + i * n_r] = ii;
//            index_copy[ii + i * n_r] = ii;
//        }
//    }
//given an array of bools n_q * n_r, see if bool true, if true start new kernel which dots the two points 
__global__ void brute_shared(int * bucket)    
{
        //Start kernel |    
}

void lsh_test(des_t *q_points, des_t *r_points, int n_q, int n_r, float4 *sorted, int nbits, int l, int max_dist)
{  
    // see how much memory we have  

    size_t free_byte ;
    size_t total_byte ;
    hipMemGetInfo( &free_byte, &total_byte ) ;

    // need at least  

    //double free_db = (double)free_byte ;
    //double total_db = (double)total_byte ;
    //double used_db = total_db - free_db ;

   // printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

   //         used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
 
    // arry of vectors 
    des_t *rand_array;
    // hash codes  
    int *code_r, *code_q;
    // index into bucket array and copy to sort 
    int *index, *index_copy ; 
    // given bucket n gives start index 
    int *bucket_start;
    // dot from random vector to q / r points 
    float * dot_res_r, * dot_res_q;
 
    // a bucket of all the points each q has to check   
    // number of buckets within hamming distance r given n bits = n!/r!(n-r)! where r is 1 -> r  
    int size_bucket = 0 ;
    for (int i = 1; i < max_dist + 1; i++)
    {
        size_bucket += fact(nbits)/(fact(i)*fact(nbits - i)) ;  
    }

    printf("need %i \n", size_bucket) ; 
    int *buckets;

    hipMallocManaged((void **)&buckets, sizeof(int) * n_q * size_bucket);
    hipMemset(buckets, 0, sizeof(int) * n_q * size_bucket);
    
    hipMallocManaged((void **)&rand_array, sizeof(des_t) * nbits);
    hipMallocManaged((void **)&index, sizeof(int) * n_r);  
    hipMallocManaged((void **)&index_copy, sizeof(int) * n_r);
    hipMallocManaged((void **)&bucket_start, (2 << nbits) * sizeof(int));

    hipMallocManaged((void **)&code_r, sizeof(int) * n_r);
    hipMallocManaged((void **)&code_q, sizeof(int) * n_q);

    hipMallocManaged((void **)&dot_res_r, nbits * n_r* sizeof(float)); 
    hipMallocManaged((void **)&dot_res_q, nbits * n_q* sizeof(float));

    for (int L = 0; L < l; L++)
    {
        // memsetstuff
        hipMemset(index, 0, sizeof(int) * n_r );
        hipMemset(index_copy, 0, sizeof(int) * n_r );
        hipMemset(code_r, 0, sizeof(int) * n_r );
        hipMemset(code_q, 0, sizeof(int) * n_q);
        // todo make kernel to set - 1  
        for (int i = 0; i < (2 << nbits); i++)
        {
            bucket_start[i] = -1;
        }
        // make random vectors
        for (int i = 0; i < nbits; i++)
        {
            make_vec(128, rand_array[i]);
        }
        // dot random vectors with n_r
        dim3 grid_dot_r(n_r, nbits, 1) ;
        dim3 block_dot_r(32, 1, 1) ;   
        dot_gpu<<<grid_dot_r, block_dot_r>>>(rand_array, r_points, dot_res_r); 
        // set bit for code_r 
        dim3 grid_bit_r(n_r,1,1) ; 
        dim3 block_bit_r(nbits,1,1) ; 
        set_bit<<<grid_bit_r, block_bit_r>>>(code_r, nbits, dot_res_r) ; 
        hipDeviceSynchronize();

        // make buckets for r points
        dim3 grid_set(n_r/32+1, 1, 1) ; 
        dim3 block_set(32,3,1) ;
        set_bucket<<<grid_set, block_set>>>(index, index_copy, n_r) ; 
        hipDeviceSynchronize();
        //sort bucket by index  
        // gpu or cpu 
        IndexCompare tc(index_copy, code_r);
    #if PREFER_CPU == 0
        thrust::device_ptr<int> ptr = thrust::device_pointer_cast(index);
        thrust::sort( ptr, ptr + n_r, tc );
    #else
        hipDeviceSynchronize();
        int* ptr = index;
        std::sort( ptr, ptr + n_r, tc );
    #endif
        // set bucket start  
        for (int i = 0; i < n_r; i++)
        {
            if (bucket_start[code_r[index[i]]] == -1)
            {
                bucket_start[code_r[index[i]]] = i;
            }
        }

        // dot random vectors with q
        dim3 grid_dot_q(n_q, nbits, 1) ;
        dim3 block_dot_q(32, 1, 1) ;   
        dot_gpu<<<grid_dot_q, block_dot_q>>>(rand_array, q_points, dot_res_q); 

        //set bit for hash values for code_q 
        dim3 grid_bit_q(n_r,1,1) ; 
        dim3 block_bit_q(nbits,1,1) ; 
        set_bit<<<grid_bit_q, block_bit_q>>>(code_q, nbits, dot_res_q) ; 
        hipDeviceSynchronize();

        // fill buckets
        // do all the dots at once insted of dividing into L then do  
        // 1 one time for each q use n_r size array so we dont get duplicates  
        // 2 same but  multpile qs at the same time 
        // have a intilzie funtoins which make the data we need  ? 
        // have all the hamming 
        dim3 grid_bucket(n_q, 1, 1) ; 
        dim3 block_bucket(max_dist,1,1) ; 
        hamming<<<grid_bucket, block_bucket>>>(buckets, max_dist, nbits, code_q ) ; 

       hipDeviceSynchronize();
       for (int ii = 0; ii < n_q; ii++)
       {
            int bucket = code_q[ii];
            int c = 0 ; 
            for (int n = 0; n < max_dist; n++)
            {
                int counters[n + 1];

                for (int q = 0; q < (n + 1); q++)
                {
                    counters[q] = q;
                }

                bool done = false;
                
                while (!done)
                {
                    int neighbour_bucket = bucket;
                    for (int nn = 0; nn < (n + 1); nn++)
                    {
                        neighbour_bucket ^= 1UL << counters[nn];
                    }
                    buckets[ii * size_bucket + c] = neighbour_bucket ; 
                    c ++ ; 
                    bool flag = false;
                    int nnn = n;
                    int bits = nbits;
                    while (!flag)
                    {

                        if (((counters[nnn] + 1) >= bits) && nnn == 0)
                        {
                            flag = true;
                            done = true;
                        }
                        else if ((counters[nnn] + 1) < bits)
                        {
                            counters[nnn] += 1;
                            flag = true;
                        }

                        nnn--;
                        bits--;
                    }
                }
            }
           // printf("int c = %i \n", c) ; 

        }
        for (int i = 0; i < n_q; i++)
        {
            if(L == 0)
            {
                sorted[i].w = MAXFLOAT;
                sorted[i].x = MAXFLOAT;
                sorted[i].y = MAXFLOAT;
                sorted[i].z = MAXFLOAT;
            }
            search_bucket(sorted[i], code_q[i], bucket_start[code_q[i]], code_r, index, r_points, q_points, n_r, i) ; 
            for (int ii = 0; ii < size_bucket; ii++)
            {
                int iii = buckets[max_dist * i + ii]  ; 
                search_bucket(sorted[i], iii, bucket_start[iii], code_r, index, r_points, q_points, n_r, i) ; 
            }
        }
    } 
}

void search_bucket(float4 &min, int bucket, int start, int * code, int * index, des_t * r_p, des_t * q_p, int size_r, int x)
{
    if(start == -1 ) return ; 
    int i = start ; 
    while (i < size_r && code[index[i]] == bucket)
    {
        float dist = host_lenght(r_p[i], q_p[x]);

        if (dist < min.x)
        {
            min.y = min.x;
            min.x = dist;

            min.w = min.z;
            min.z = i;
        }
        else
        {
            if (dist < min.y)
            {
                min.y = dist;
                min.w = i;
            }
        }
        i ++ ; 
    }
}