#include "hip/hip_runtime.h"
#include <iostream>
#include "hiprand/hiprand_kernel.h"
#include <string>
#include "hipblas.h"
#include "lsh.h"
#include "knn_brute.h"
#include <memory>
#include "helper.h"
#include <vector>
#include <algorithm>
#include <bits/stdc++.h>

using namespace std;

void host_lsh(des_t *q_points, des_t *r_points, int n_q, int n_r, float4 *sorted, int nbits, int l, int max_dist)
{
    des_t *rand_array;
    hipMallocManaged((void **)&rand_array, sizeof(des_t) * nbits * l);
    // make random vectors
    for (size_t i = 0; i < l * nbits; i++)
    {
        make_vec(128, rand_array[i]);
    }
    // make an array of ints with one int for each r_point
    int *code;
    int * code_test ; 
    int *index;
    // need a copy to sort using sort
    int *index_copy;
    int *bucket_start;

    int * index_test ; 
    int * index_copy_test ; 
    hipMallocManaged((void **)&index, sizeof(int) * n_r * l);
    hipMemset(index, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&index_copy, sizeof(int) * n_r * l);
    hipMemset(index_copy, 0, sizeof(int) * n_r * l);
    
    hipMallocManaged((void **)&index_test, sizeof(int) * n_r * l);
    hipMemset(index_test, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&index_copy_test, sizeof(int) * n_r * l);
    hipMemset(index_copy_test, 0, sizeof(int) * n_r * l);


    hipMallocManaged((void **)&bucket_start, (2 << nbits) * sizeof(int));

    for (int i = 0; i < (2 << nbits); i++)
    {
        bucket_start[i] = -1;
    }

    hipMallocManaged((void **)&code, sizeof(int) * n_r * l);
    hipMemset(code, 0, sizeof(int) * n_r * l);
    hipMallocManaged((void **)&code_test, sizeof(int) * n_r * l);
    hipMemset(code_test, 0, sizeof(int) * n_r * l);


    // test if our gpu dot works 
    float * dot_res;
    hipMallocManaged((void **)&dot_res, l * nbits * n_r* sizeof(float));
    double s = start_timer() ; 
    dim3 grid(n_r, nbits, l) ;
    dim3 block(32, 1, 1) ;   
    dot_gpu<<<grid, block>>>(rand_array, r_points, dot_res); 
    hipDeviceSynchronize();
    grid.z = 1 ; 
    grid.y = l ; 
    block.x = nbits ;
    set_bit<<<grid, block>>>(code_test, nbits, dot_res) ; 
    hipDeviceSynchronize();
    print_time(s, "gpu") ; 
    s = start_timer() ; 
    for (int i = 0; i < n_r; i++)
    {
        for (int ii = 0; ii < l; ii++)
        {
            for (int iii = 0; iii < nbits ; iii++)
            {
                float sum = dot(r_points[i] ,rand_array[iii + ii * nbits] );
          //     printf("sum = %f ", sum) ; 
          //     printf("gpu = %f \n", dot_res[i * nbits * l + ii * nbits + iii]) ; 
                if (sum <= 0)
                {
                    code[i * l + ii ] |= 1UL << iii;
                }

            }
                if(code[i * l + ii] != code_test[i * l + ii])
                {
                    printf("cpu = %i ", code[i * l + ii]) ; 
                    printf("gpu = %i \n", code_test[i*l + ii] ) ;                       
 
                }
       }
    }
    print_time(s, "cpu") ; 
     
    
 //   // dot all vectors and add the bit to the coresponding int bit for the r points
 //   for (int i = 0; i < l; i++)
 //   {
 //       for (int ii = 0; ii < n_r; ii++)
 //       {
 //           //            printf(" %i bucket = ", (ii + i * n_r));
 //           for (int iii = 0; iii < nbits; iii++)
 //           {
 //               float sum = dot(r_points[ii], rand_array[iii + i * nbits]);
 //               if (sum <= 0)
 //               {
 //                   code[ii + i * n_r] |= 1UL << iii;
 //               }
 //           }
 //           //           printf(" %i ", code[ii + i * n_r]);
 //           //           printf("\n \n");
 //       }
 //   }

    // make buckets for r points
    grid.x = n_r/32 +1 ; 
    grid.y = l ; 
    block.x = 32 ; 
    block.y = 3 ; 
    s = start_timer() ; 
    set_bucket<<<grid,block>>>(index_test, index_copy_test, n_r) ; 
    print_time(s, "gpu"); 
    hipDeviceSynchronize();
    s = start_timer() ; 
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_r; ii++)
        {
            index[ii + i * n_r] = ii;
            index_copy[ii + i * n_r] = ii;
        }
    }

    print_time(s, "cpu") ; 
    std::sort(index, index + n_r * l, [&](const int &i, const int &j) -> bool
              { return (code[index_copy[i]] < code[index_copy[j]]); });

    //  set_bucket_start<<<grid, block>>>(helper, bucket_start, l, n_r);
    //  hipDeviceSynchronize();
    for (int i = 0; i < n_r * l; i++)
    {
        if (bucket_start[code[index[i]]] == -1)
        {
            bucket_start[code[index[i]]] = i;
        }
    }

    int *code_q;
    hipMallocManaged((void **)&code_q, sizeof(int) * n_r * l);
    hipMemset(code_q, 0, sizeof(int) * n_r * l);
    // dot all q points
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_q; ii++)
        {
            for (int iii = 0; iii < nbits; iii++)
            {
                float sum = dot(q_points[ii], rand_array[iii + i * nbits]);
                if (sum <= 0)
                {
                    code_q[ii + i * n_q] |= 1UL << iii;
                }
            }
        }
    }

    int *buckets;
    hipMallocManaged((void **)&buckets, sizeof(int) * n_q * n_r);
    hipMemset(buckets, 0, sizeof(int) * n_q * n_r);
    // fill buckets
    for (int i = 0; i < l; i++)
    {
        for (int ii = 0; ii < n_q; ii++)
        {
            int bucket = code_q[ii + (i * n_q)];
            int start = bucket_start[bucket];
            int iii = start;
            while ((start != -1) && code[index[iii]] == bucket)
            {
                buckets[ii * n_r + index[iii]] = 1;
                iii++;
            }

            // add from negbouring buckets up to n bits away // n can be given by input but is by defualt 1
            // is there any meaing to adding more than

            // make all buckets with a hamming distance of n

            // 0000
            // 1000 0100 0010 0001
            // 1100 1010 1001
            // 0110 0101
            // 0011

            // 010
            // 011 000 110
            // 001 100
            for (int n = 0; n < max_dist; n++)
            {
                int counters[n + 1];

                for (int q = 0; q < (n + 1); q++)
                {
                    counters[q] = q;
                }

                bool done = false;
                while (!done)
                {
                    int neighbour_bucket = bucket;
                    for (int nn = 0; nn < (n + 1); nn++)
                    {
                        neighbour_bucket ^= 1UL << counters[nn];
                    }
                    // printf("bucket is %i neighbour is %i \n", bucket, neighbour_bucket) ;
                    // we have bucket
                    int start = bucket_start[neighbour_bucket];
                    int iii = start;
                    while ((start != -1) && code[index[iii]] == neighbour_bucket)
                    {
                        buckets[ii * n_r + index[iii]] = 1;
                        iii++;
                    }
                    bool flag = false;
                    int nnn = n;
                    int bits = nbits;
                    while (!flag)
                    {

                        if (((counters[nnn] + 1) >= bits) && nnn == 0)
                        {
                            flag = true;
                            done = true;
                        }
                        else if ((counters[nnn] + 1) < bits)
                        {
                            counters[nnn] += 1;
                            flag = true;
                        }

                        nnn--;
                        bits--;
                    }
                }
            }
            //         printf("\n");
        }
    }

    for (int i = 0; i < n_q; i++)
    {
        sorted[i].w = MAXFLOAT;
        sorted[i].x = MAXFLOAT;
        sorted[i].y = MAXFLOAT;
        sorted[i].z = MAXFLOAT;
        for (int ii = 0; ii < n_r; ii++)
        {
            if (buckets[n_r * i + ii] == 1)
            {
                float dist = host_lenght(r_points[ii], q_points[i]);

                if (dist < sorted[i].x)
                {
                    sorted[i].y = sorted[i].x;
                    sorted[i].x = dist;

                    sorted[i].w = sorted[i].z;
                    sorted[i].z = ii;
                }
                else
                {
                    if (dist < sorted[i].y)
                    {
                        sorted[i].y = dist;
                        sorted[i].w = ii;
                    }
                }
            }
        }
    }
}

void make_vec(int dim, des_t &vec)
{
    for (size_t i = 0; i < dim; i++)
    {
        vec[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
    }
}

float dot(des_t v1, des_t v2)
{

    float sum = 0.f;
    for (size_t i = 0; i < 128; i++)
    {
        float a = ((float *)v1)[i] -0.5;
        float b = ((float *)v2)[i]- 0.5;
        float c = a * b;
        sum += c;
    }
    return sum;
}
float test_dot(float *v1, des_t v2)
{
    float sum = 0.f;
    for (size_t i = 0; i < 128; i++)
    {
        float a = ((float *)v1)[(4 * 4) * i];
        float b = ((float *)v2)[i];
        float c = a * b;
        sum += c;
    }
    return sum;
}
void gpu_lsh(des_t *q_points, des_t *r_points, int n_q, int n_r, float4 *sorted, int nbits, int l, int max_dist)
{
    // make random vectors

    des_t *rand_array;
    hipMallocManaged((void **)&rand_array, sizeof(des_t) * nbits * l);

    // fill array
    uint64_t seed = clock();
    dim3 grid_size(nbits * l, 1, 1);
    dim3 block_size(128, 1, 1);

    //fill in the rand array
    random_vector<<<grid_size, block_size>>>(seed, rand_array);

    // make an array of ints with one int for each r_point
    int *code;
    int *index;
    // need a copy to sort using sort
    int *index_copy;
    int *bucket_start;

    hipMallocManaged((void **)&index, sizeof(int) * n_r * l);
    hipMemset(index, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&index_copy, sizeof(int) * n_r * l);
    hipMemset(index_copy, 0, sizeof(int) * n_r * l);

    hipMallocManaged((void **)&bucket_start, (2 << nbits) * sizeof(int));

    // dot rand_array and r_points to make buckets  


    // cublas can maybe be used if the query is big enough how big test todo   
    // dot_res layout [x][l * nbits]
    float *dot_res;
    hipMallocManaged((void **)&dot_res, l * nbits * n_r);

   // float a = 1.0f;
   // float b = 1.0f;
   // hipblasHandle_t handle;
   // hipblasCreate(&handle);

   // hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, l * nbits, n_r, 128, &a, (float *)rand_array, l * nbits, (float *)r_points, 128, &b, dot_res, l * nbits);
   // 
    /**
     * rand_arraay = L*nbitsX128
     * r_points = 128 X n_r
     * dot_res = L*nbits X n_r if we were to use cublas  so n_r * L*nbits  
     * dot_res[] 
     **/
  // test there are some difrencece caused by the way the calculation is done  
  //  for (int i = 0; i < n_r; i++)
  //  {
  //      for (int ii = 0; ii < (l * nbits); ii++)
  //      {
  //          compare_float(test_dot((rand_array + ii), r_points[i]), dot_res[i * (l * nbits) + ii]);
  //      }
  //  }


    dim3 grid(n_r, l, 1) ;
    dim3 block(32, nbits, 1) ;   
    dot_gpu<<<grid, block>>>(rand_array, r_points, dot_res); 

    
    //index
    // code[index[r number ]] -> bucket
    //

    // index[range (bucket start[bucket] ->  [while code [index in r:points]] == bucket)] -> index in r_points of elemtn in bucket
    // r_points[index in r_points ] -> dest vec
//
//    int *helper;
//    hipMallocManaged((void **)&helper, sizeof(int) * n_r * l);
//
//    //set bucket start
//    dim3 block_size(32, 1, 1);
//    dim3 grid_size(((l * n_r) / 32) + 1, 1, 1);
//    set_helper<<<grid_size, block_size>>>(helper, code, index);
//
//    hipDeviceSynchronize();
//    dim3 block(32, 1, 1);
//    dim3 grid(n_r * l, 1, 1);
}

// kernels for finding the start of each bucket in the index array
// not sure if this is really faster than cpu todo TEST
// sets helper values
__global__ void set_helper(int *helper, int *code, int *index)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    helper[idx] = code[index[idx]];
}

__global__ void set_bucket_start(int *helper, int *bucket_start, int l, int n_r)
{
    for (int i = threadIdx.x; i < (l * n_r); i += 32)
    {

        int v = (helper[i] == blockIdx.x);
        unsigned int mask = __ballot_sync(0xffffffff, v);

        if (__popc(mask) == 0)
            continue;
        int x = __ffs(mask);
        if (threadIdx.x == 0)
        {
            int g = i + x - 1;
            bucket_start[blockIdx.x] = g;
            break;
        }
    }
}
// n_r, l, 1 
// nbits ,1 ,1 for now we use 32 threads but could also use 16 8 4 2 or 64 / 128  
// set the bit for the one random array then combine whitin warp to form 32 bit buckets 
// want buckets to have layaout 
// bucket[0][0] ......... bucket[0][l]
// bucket[n][0] --------- bucket[n][l]
//
__global__ void set_bit(int *buckets, int nbits, float * dot)
{
    int var = 0 ; 
    // index of the dot prouduct we need 
    
    int dot_idx =  blockIdx.x * gridDim.y * blockDim.x  + blockIdx.y * blockDim.x + threadIdx.x ;    
    // only care if its a relevent thread 
    if((threadIdx.x ) < nbits )
    {
        if(dot[dot_idx] <= 0 ) 
        {
            var |= 1UL << threadIdx.x;
        }
    }
    var += __shfl_down_sync( 0xffffffff, var, 16 );
    var += __shfl_down_sync( 0xffffffff, var, 8 ); 
    var += __shfl_down_sync( 0xffffffff, var, 4 ); 
    var += __shfl_down_sync( 0xffffffff, var, 2 );
    var += __shfl_down_sync( 0xffffffff, var, 1 );   
    if(threadIdx.x == 0)
    {
        buckets[blockIdx.x * gridDim.y + blockIdx.y ] = var ;     
    }  
}
//want dot array to be 
// point 0 * rand [0][0 - nbits], .......   point n * rand [n][0 - nbits]
// point n * rand [0][0 - nbits], .......   point n * rand [n][0 - nbits] 
// 
// want rand[n] to only be read by blocks on the same sm. load in shared memory  
//  or want points[n ] to only be read by blocks on the same sm 
__global__ void dot_gpu(des_t *  rand, des_t * points, float *dot)
{
    // called with 
    // n, nbits, l grid  
    //32 1 1 block 
    // could change to 32, x ,1 block todo test if faster 0_0 
    
    float res = 0.f ; 
    float4 a = ((float4 * )points[blockIdx.x])[threadIdx.x ];
    float4 b = ((float4 * )rand[blockIdx.z * gridDim.y + blockIdx.y])[threadIdx.x]; 

    res +=
        (a.x -0.5)*(b.x - 0.5) + (a.y -0.5)*(b.y -0.5) +
        (a.z -0.5)*(b.z - 0.5) + (a.w -0.5)*(b.w -0.5)  ;  
    reduce(res) ; 
    if(threadIdx.x == 0)
    {
        dot[blockIdx.x * gridDim.y * gridDim.z + blockIdx.z * gridDim.y + blockIdx.y] = res ;     
    }  
}


// initialize array to a value
__global__ void initialize(int *array, int value)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    array[idx] = value;
}

// may have to make vectors more random ! hmm todo
// makes a random float

__device__ inline float random_float(uint64_t seed, int idx)
{
    hiprandState s;
    hiprand_init(seed + idx, 0, 0, &s);
    return hiprand_uniform(&s);
}
//fills a vector with random floats
__global__ void random_vector(uint64_t seed, des_t *array)
{
    int idx = threadIdx.x;
    float * vec = (float *)array[blockIdx.x]  ;
    vec[idx] = random_float(seed, idx);
    
}
__device__ inline void reduce(float &var)
{
    var += __shfl_down_sync( 0xffffffff, var, 16 );
    var += __shfl_down_sync( 0xffffffff, var, 8 ); 
    var += __shfl_down_sync( 0xffffffff, var, 4 ); 
    var += __shfl_down_sync( 0xffffffff, var, 2 );
    var += __shfl_down_sync( 0xffffffff, var, 1 );   
}
// called with 
// grid n_r/32 +1 l 1
// block 32 3 1 

__global__ void set_bucket(int * index, int * index_copy, int n)
{
    int i = blockDim.x * blockDim.y * blockIdx.x +  blockDim.x * threadIdx.y + threadIdx.x ; 
    if(n > i)
    {
        index[i + blockIdx.y * n] = i ; 
        index_copy[i + blockIdx.y * n] = i ; 
    }
}
//    for (int i = 0; i < l; i++)
//    {
//        for (int ii = 0; ii < n_r; ii++)
//        {
//            index[ii + i * n_r] = ii;
//            index_copy[ii + i * n_r] = ii;
//        }
//    }

