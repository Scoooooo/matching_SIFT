#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "knn_brute.h"
#include <string>
#include "hipblas.h"
#include "lsh.h"
#include "knn_brute.h"
#include "helper.h"
#include <algorithm>

// thrust 
#include <thrust/reduce.h>
#include <thrust/gather.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/fill.h>
// gpu brute force 2nn 
// takes pointer with data on device as input, sorted output should also be on devcie or just manged 
struct abs2 {
    __host__ __device__ float operator()(const float &x) const { return x * x; }
};

__global__ void assemble_final_result(const float * __restrict__ d_norms_x_2, const float * __restrict__ d_norms_y_2, float * __restrict__ d_dots,
                                      const int NX, const int NY) {

    const int i = threadIdx.x + blockIdx.x * gridDim.x;
    const int j = threadIdx.y + blockIdx.y * gridDim.y;

    if ((i < NY) && (j < NX)) d_dots[i * NX+ j] = d_norms_x_2[j] + d_norms_y_2[i] - 2 * d_dots[i * NX+ j];

}

void cublas_2nn_brute(des_t * q_points, des_t * r_points, int q_n, int r_n, float4  * sorted)
{
// steps ||x,y||^2 = ||x||^2 + ||y||^2 - 2x*y^T
// notes cublas works in colum major,, c++ is in row major :(  
// meaning our input q_points and r_poins are already transpoed 
// first we need to get the norms of x and y 
// then we need to multiply every element with itself 
// then we need to 
    const int Ndims = 128;        // --- Number of rows
    const int NQ    = q_n;        // --- Number of columns
    const int NR    = r_n;        // --- Number of columns

    //cublas
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float a = 1.f;
    float b = 0.f;
 
    
    // norm of q 
   // thrust::device_vector<float> q_N(NX);
    float * q_N; 
    hipMallocManaged((void **)&q_N, q_n * sizeof(float)) ; 

    thrust::device_vector<float> q_2(Ndims * NQ);
    thrust::transform((float * )q_points, (float * )q_points + q_n, q_2.begin(), abs2());
    thrust::device_vector<float> d_ones(Ndims, 1.f);

    hipblasSgemv(handle, HIPBLAS_OP_N, Ndims, NQ, &a, thrust::raw_pointer_cast(q_2.data()), Ndims, 
                               thrust::raw_pointer_cast(d_ones.data()), 1, &b, q_N, 1);
   // hipblasSgemv(handle, HIPBLAS_OP_N, 1, q_n, 128, &a, thrust::raw_pointer_cast(d_ones.data()), 1, (float *)q_points, 128, &b,  q_N,1 );

    // norm of r 
//    thrust::device_vector<float> r_N(NX);
    float * r_N; 
    hipMallocManaged((void **)&r_N, r_n * sizeof(float)) ; 


    thrust::device_vector<float> r_2(Ndims * NR);
    thrust::transform((float * )r_points, (float * )r_points + r_n, r_2.begin(), abs2());

    hipblasSgemv(handle, HIPBLAS_OP_N, Ndims, NR, &a, thrust::raw_pointer_cast(r_2.data()), Ndims, 
                               thrust::raw_pointer_cast(d_ones.data()), 1, &b, r_N, 1);
    //hipblasSgemv(handle,  HIPBLAS_OP_N, 1, r_n, 128, &a, thrust::raw_pointer_cast(d_ones.data()), 1, (float *)r_points, 128, &b,  r_N,1 );
//
    /***********************************/
    /* CALCULATING THE SCALAR PRODUCTS */
    /***********************************/
//  thrust::device_vector<float> d_dots(NX * NY);
// array of the distances between all q and r points 
    float * dev_dist_f ; 
    hipMallocManaged((void **)&dev_dist_f, q_n* r_n * sizeof(float)) ; 
//
//
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, NQ, NR, Ndims, &a,
                               (float * )q_points, Ndims, (float * )r_points, Ndims, &b,
                               dev_dist_f, NQ);
    float * dev_dist ; 
    //array of dist from each q point to every r point 
    hipMallocManaged((void **)&dev_dist, q_n* r_n * sizeof(float)) ; 

    dim3 grid_size(q_n, r_n, 1) ;
    dim3 block_size(32, 1, 1) ;   

    //fill in the dist array
    sqrEuclidianDist<<<grid_size, block_size>>>(q_points,r_points, dev_dist);
    
    hipDeviceSynchronize() ; 
    for (size_t i = 0; i < 10; i++)
    {

        printf("%f \n", r_N[i]); 
        printf("%f \n", q_N[i]); 
        printf("%f \n", dev_dist_f[i]); 
        printf("%f == %f\n", dev_dist[i], r_N[i] + q_N[i] - 2*dev_dist_f[i]); 
    }
    


    hipFree(r_N); 
    hipFree(q_N); 
    hipFree(dev_dist_f); 
}

void device_brute(des_t * q_points, des_t * r_points, int q_n, int r_n, float4  * sorted)
{
    // array of the distances between all q and r points 
    float * dev_dist ; 
    //array of dist from each q point to every r point 
    hipMallocManaged((void **)&dev_dist, q_n* r_n * sizeof(float)) ; 

    dim3 grid_size(q_n, r_n, 1) ;
    dim3 block_size(32, 1, 1) ;   

    //fill in the dist array
    sqrEuclidianDist<<<grid_size, block_size>>>(q_points,r_points, dev_dist);
    hipDeviceSynchronize();

    dim3 gridSize(q_n,1,1) ;
    dim3 blockSize(32,1,1) ; 

    min_dist<<<gridSize,blockSize>>>(dev_dist, r_n , sorted) ; 
    hipDeviceSynchronize();
    hipFree(dev_dist) ; 
}

//kernels
//finds the sqr euclidan distance between two 128 vector arrays
__global__ void sqrEuclidianDist(des_t * q_points, des_t * r_points, float * dist_array)   
{   
    float dist = 0.0f ;
    // find dist 
    for (size_t i = 0; i < 4; i++)
    {
        float a = ((float *)q_points[blockIdx.x])[threadIdx.x+(i*32)]; 
        float b = ((float *)r_points[blockIdx.y])[threadIdx.x+(i*32)]; 
        float c = a - b ; 
        dist += c * c ; 
    }

    dist += __shfl_down_sync( 0xffffffff, dist, 16 );
    dist += __shfl_down_sync( 0xffffffff, dist, 8 ); 
    dist += __shfl_down_sync( 0xffffffff, dist, 4 ); 
    dist += __shfl_down_sync( 0xffffffff, dist, 2 );
    dist += __shfl_down_sync( 0xffffffff, dist, 1 );   
    if(threadIdx.x == 0)
    {
        dist_array[blockIdx.x * gridDim.y + blockIdx.y] = dist; 
    }
}
//find smallest vlaue in the warp and index  
__device__ inline void best_in_warp(float4  &min_2)
{    
    for (int i = 16; i > 0; i/= 2)
    {          
        float x_dist = __shfl_down_sync( 0xffffffff, min_2.x, i );
        float y_dist = __shfl_down_sync( 0xffffffff, min_2.y, i );
        float w_value = __shfl_down_sync( 0xffffffff, min_2.w, i );
        float z_value = __shfl_down_sync( 0xffffffff, min_2.z, i );
        if(x_dist < min_2.x)
        {
            min_2.y = min_2.x ; 
            min_2.x = x_dist ;  
                
            min_2.w = min_2.z ; 
            min_2.z = z_value;  
        }
        else{
            if(x_dist < min_2.y)
            {
                min_2.y = x_dist ; 
                min_2.w = z_value;  
                continue ; 
            }
        } 
        if(y_dist < min_2.y)
        {
            min_2.y = y_dist ; 
            min_2.w = w_value;  
        }
    }
}
 
// x warps per dist 
__global__ void min_dist(float *  dist, int size ,float4 * sorted)
{
    //           finds the dist array         x dim       y dim pos                      
    int offset = (blockIdx.x * size)+ threadIdx.y * blockDim.x ;
   
    float4 min_2 ;  
    min_2.x = MAXFLOAT; 
    min_2.y = MAXFLOAT; 

    for (int i = 0; (i + threadIdx.x +  threadIdx.y * blockDim.x  ) < size ; i+=(blockDim.x * blockDim.y) )
    {
        if(dist[i + offset + threadIdx.x] < min_2.x)
        {
            min_2.y = min_2.x ; 
            min_2.x = dist[i + offset + threadIdx.x ] ;  
            min_2.w = min_2.z ;  
            min_2.z = i + threadIdx.x + threadIdx.y * blockDim.x  ; 
        }
        else{
            if(dist[i + offset + threadIdx.x ] < min_2.y)
            {
                min_2.y = dist[i + offset + threadIdx.x ] ;  
                min_2.w = i + threadIdx.x + threadIdx.y * blockDim.x   ;  
            }
        }
    }
    best_in_warp(min_2) ;   

    __shared__ float4 best[32] ; 
    
    if(threadIdx.y == 0)
    {
        best[threadIdx.x].x = MAXFLOAT ;
        best[threadIdx.x].y = MAXFLOAT ; 
    }
    __syncthreads() ; 
    if(threadIdx.x == 0)
    {
        best[threadIdx.y] = min_2 ;
    }
    
    __syncthreads() ; 
    if(threadIdx.y == 0){

        min_2 = best[threadIdx.x] ; 
        best_in_warp(min_2) ; 
        if(threadIdx.x == 0)
        { 
            sorted[blockIdx.x ] = min_2 ; 
        }
    }
}

//host brute

void host_brute(des_t * q_points, des_t * r_points, int q_points_size, int r_points_size, float4  * sorted)
{
    
    float * lenght;
    hipHostMalloc((void **)&lenght, r_points_size * q_points_size * sizeof(float)) ; 
    for (size_t i = 0; i < q_points_size; i++)
    {
        for (size_t ii = 0; ii < r_points_size; ii++)
        {
            lenght[(i * r_points_size) + ii ] = host_lenght(q_points[i], r_points[ii]) ;  
        }
    }
    
    host_sort(lenght,r_points_size, q_points_size, sorted) ; 
    hipFree(lenght) ; 
}

//given an array of arrays of lenghts it sorts the array and returns a sroted array 
//containing the 2 shortests lenghts from each array 
void host_sort(float * dist, int size, int array_size, float4 * sorted)
{   
    for (int i = 0; i < array_size ; i++)
    {
        float4 min_2 ;  
        min_2.x = MAXFLOAT ; 
        min_2.y = MAXFLOAT ;
        min_2.z = MAXFLOAT ;  
        min_2.w = MAXFLOAT ;  
        int offset = i * size ; 
        for (int ii = 0; ii < size; ii++)
        {
            if(dist[ii + offset] < min_2.x)
            {
                min_2.y = min_2.x ; 
                min_2.x = dist[ii + offset] ;  
                
                min_2.w = min_2.z ; 
                min_2.z = ii ;  
                
            }
            else{
                if(dist[ii + offset] < min_2.y)
                {
                    min_2.y = dist[ii + offset] ; 
                    min_2.w = ii ;  
                }
            }
        }
        sorted[i] = min_2 ;  
    }
}

// given 2 points find the euclidina lenght before taking the root
float host_lenght(des_t x, des_t y){
    float * vec1 = (float * )x ; 
    float * vec2 = (float * )y ;
    float dist =  0.0f  ;  
    for (size_t i = 0; i < 128; i++)
    {
        float a = vec1[i] ; 
        float b = vec2[i] ;  
        float c = a - b ; 
        dist += c * c ; 
    }
    return dist ; 
}



