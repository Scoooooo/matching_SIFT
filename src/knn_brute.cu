#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "knn_brute.h"
#include <string>
#include "hipblas.h"
#include "lsh.h"
#include "knn_brute.h"
#include "helper.h"
#include <algorithm>

// thrust 
#include <thrust/reduce.h>
#include <thrust/gather.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/pair.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/fill.h>

// makes sure that we have enough memory  
void cublas_2nn_f(des_t_f * q_points, des_t_f * r_points, int q_n, int r_n, float4  * sorted, hipblasHandle_t handle)
{
    size_t free_byte ;
    size_t total_byte ;
    hipMemGetInfo( &free_byte, &total_byte ) ;

    double free_db = (double)free_byte ;
    double total_db = (double)total_byte ;
    double used_db = total_db - free_db ;

    printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
     used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

    int i = 1 ; 
    size_t need_byte = (size_t)q_n *  (size_t)r_n * 4 ;  

    int temp = q_n ; 
    size_t dont_use = 10000000000 ; 
    while ( need_byte > (free_byte - dont_use ))
    {
        printf("%i, %zu, %zu is \n", i, need_byte, free_byte - dont_use) ; 
        i ++ ; 
        temp = q_n / i ; 
        need_byte = (size_t)temp* (size_t) r_n * 4 ;  
    }
    printf("%i, %zu, %zu is \n", i, need_byte, free_byte - dont_use) ; 
    float * dist ; 
    hipMalloc((void **)&dist, (size_t)temp*  (size_t)r_n * 4) ; 
    int ii; 
    for (ii = 0; ii < i; ii++)
    {
        cublas_2nn_brute_f(q_points + (ii * temp), r_points, temp, r_n, sorted + (ii * temp), dist, handle); 
        printf("%i \n", ii) ; 
    }
    if((q_n % temp ) > 0 )
    {
        int left =  q_n % temp ; 
        printf("left %i \n", left) ; 
        cublas_2nn_brute_f(q_points + (ii * temp), r_points, left, r_n, sorted + (ii * temp), dist, handle); 
    }

    hipFree(dist); 
}
// gpu brute force 2nn 
// takes pointer with data on device as input, sorted output should also be on devcie or just manged 
void cublas_2nn_brute_f(des_t_f * q_points, des_t_f * r_points, int q_n, int r_n, float4  * sorted, float * dist,hipblasHandle_t handle)
{
// steps d(x,y)^2 = ||x||^2 + ||y||^2 - 2x*y^T
// for sift we only need to solve d(x,y)^2 = 2 + -2x*y^t
// or -x*y^t
// notes cublas works in colum major,, c++ is in row major :(  
// meaning our input q_points and r_poins are already transpoed 
    // add minus 
    float a = -1.f;
    float b = 0.f;
 
   // we are in row major so we want our output from cublas to be in row major as well 
    // d^t = r * q^t is what cublas dose if see from cloum major
    // which is d = r^t * q   
    
    hipblasStatus_t stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, r_n, q_n, 128, &a, (float *)r_points, 128, (float *)q_points, 128, &b, dist, r_n);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("dot failed, cublas 2nn \n");
        hipFree (dist);
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    } 
    // want to find min value for each dist array 
    dim3 gridSize(q_n,1,1) ;
    dim3 blockSize(32,4,1) ; 
    min_dist_f<<<gridSize,blockSize>>>(dist, r_n , sorted) ; 
    hipError_t cudaStat = hipDeviceSynchronize();

    if (cudaStat != hipSuccess) {
        printf ("min dist failed, cublas 2nn \n");
        hipFree (dist);
        hipblasDestroy(handle);
        exit(EXIT_FAILURE);
    }
}

void device_brute(des_t_f * q_points, des_t_f * r_points, int q_n, int r_n, float4  * sorted)
{
    // array of the distances between all q and r points 
    float * dev_dist ; 
    //array of dist from each q point to every r point 
    hipMalloc((void **)&dev_dist, q_n* r_n * sizeof(float)) ; 

    dim3 grid_size(q_n, r_n, 1) ;
    dim3 block_size(32, 1, 1) ;   

    //fill in the dist array
    sqrEuclidianDist<<<grid_size, block_size>>>(q_points,r_points, dev_dist);
    hipDeviceSynchronize();

    dim3 gridSize(q_n,1,1) ;
    dim3 blockSize(32,1,1) ; 

    min_dist_f<<<gridSize,blockSize>>>(dev_dist, r_n , sorted) ; 
    hipDeviceSynchronize();

    hipFree(dev_dist) ; 
}

//kernels
//finds the sqr euclidan distance between two 128 vector arrays
__global__ void sqrEuclidianDist(des_t_f * q_points, des_t_f * r_points, float * dist_array)   
{   
    float dist = 0.0f ;
    // find dist 
    for (size_t i = 0; i < 4; i++)
    {
        float a = ((float *)q_points[blockIdx.x])[threadIdx.x+(i*32)]; 
        float b = ((float *)r_points[blockIdx.y])[threadIdx.x+(i*32)]; 
        float c = a - b ; 
        dist += c * c ; 
    }

    dist += __shfl_down_sync( 0xffffffff, dist, 16 );
    dist += __shfl_down_sync( 0xffffffff, dist, 8 ); 
    dist += __shfl_down_sync( 0xffffffff, dist, 4 ); 
    dist += __shfl_down_sync( 0xffffffff, dist, 2 );
    dist += __shfl_down_sync( 0xffffffff, dist, 1 );   
    if(threadIdx.x == 0)
    {
        dist_array[blockIdx.x * gridDim.y + blockIdx.y] = dist; 
    }
}
//find smallest vlaue in the warp and index  
__device__ inline void best_in_warp(float4  &min_2)
{    
    for (int i = 16; i > 0; i/= 2)
    {          
        float x_dist = __shfl_down_sync( 0xffffffff, min_2.x, i );
        float y_dist = __shfl_down_sync( 0xffffffff, min_2.y, i );
        float w_value = __shfl_down_sync( 0xffffffff, min_2.w, i );
        float z_value = __shfl_down_sync( 0xffffffff, min_2.z, i );
        if(x_dist < min_2.x)
        {
            min_2.y = min_2.x ; 
            min_2.x = x_dist ;  
                
            min_2.w = min_2.z ; 
            min_2.z = z_value;  
        }
        else{
            if(x_dist < min_2.y)
            {
                min_2.y = x_dist ; 
                min_2.w = z_value;  
                continue ; 
            }
        } 
        if(y_dist < min_2.y)
        {
            min_2.y = y_dist ; 
            min_2.w = w_value;  
        }
    }
}
 
// x warps per dist 
__global__ void min_dist_f(float *  dist, int size ,float4 * sorted)
{
    //            finds the dist array         x dim       y dim pos                      
    int offset = (blockIdx.x * size)+ threadIdx.y * blockDim.x ;
   
    float4 min_2 ;  
    min_2.x = MAXFLOAT; 
    min_2.y = MAXFLOAT; 
    //maybe better to read values first, however compiler may just be doing it for me
    for (int i = 0; (i + threadIdx.x +  threadIdx.y * blockDim.x  ) < size ; i+=(blockDim.x * blockDim.y) )
    {

        float temp = dist[i + offset + threadIdx.x]; 
        if(temp < min_2.x)
        {
            min_2.y = min_2.x ; 
            min_2.x = temp ;  
            min_2.w = min_2.z ;  
            min_2.z = i + threadIdx.x + threadIdx.y * blockDim.x  ; 
        }
        else{
            if(temp< min_2.y)
            {
                min_2.y = temp;  
                min_2.w = i + threadIdx.x + threadIdx.y * blockDim.x   ;  
            }
        }
    }
    best_in_warp(min_2) ;   

    __shared__ float4 best[32] ; 
    
    if(threadIdx.y == 0)
    {
        best[threadIdx.x].x = MAXFLOAT ;
        best[threadIdx.x].y = MAXFLOAT ; 
    }
    __syncthreads() ; 
    if(threadIdx.x == 0)
    {
        best[threadIdx.y] = min_2 ;
    }
    
    __syncthreads() ; 
    if(threadIdx.y == 0){
        min_2 = best[threadIdx.x] ; 
        best_in_warp(min_2) ; 
        if(threadIdx.x == 0)
        { 
            sorted[blockIdx.x ] = min_2 ; 
        }
    }
}

//host brute

void host_brute(des_t_f * q_points, des_t_f * r_points, int q_points_size, int r_points_size, float4  * sorted)
{
    
    float * lenght;
    hipHostMalloc((void **)&lenght, r_points_size * q_points_size * sizeof(float)) ; 
    for (size_t i = 0; i < q_points_size; i++)
    {
        for (size_t ii = 0; ii < r_points_size; ii++)
        {
            lenght[(i * r_points_size) + ii ] = host_lenght(q_points[i], r_points[ii]) ;  
        }
    }
    
    host_sort(lenght,r_points_size, q_points_size, sorted) ; 
    hipFree(lenght) ; 
}

//given an array of arrays of lenghts it sorts the array and returns a sroted array 
//containing the 2 shortests lenghts from each array 
void host_sort(float * dist, int size, int array_size, float4 * sorted)
{   
    for (int i = 0; i < array_size ; i++)
    {
        float4 min_2 ;  
        min_2.x = MAXFLOAT ; 
        min_2.y = MAXFLOAT ;
        min_2.z = MAXFLOAT ;  
        min_2.w = MAXFLOAT ;  
        int offset = i * size ; 
        for (int ii = 0; ii < size; ii++)
        {
            if(dist[ii + offset] < min_2.x)
            {
                min_2.y = min_2.x ; 
                min_2.x = dist[ii + offset] ;  
                
                min_2.w = min_2.z ; 
                min_2.z = ii ;  
                
            }
            else{
                if(dist[ii + offset] < min_2.y)
                {
                    min_2.y = dist[ii + offset] ; 
                    min_2.w = ii ;  
                }
            }
        }
        sorted[i] = min_2 ;  
    }
}

// given 2 points find the euclidina lenght before taking the root
float host_lenght(des_t_f x, des_t_f y){
    float * vec1 = (float * )x ; 
    float * vec2 = (float * )y ;
    float dist =  0.0f  ;  
    for (size_t i = 0; i < 128; i++)
    {
        float a = vec1[i] ; 
        float b = vec2[i] ;  
        float c = a - b ; 
        dist += c * c ; 
    }
    return dist ; 
}



